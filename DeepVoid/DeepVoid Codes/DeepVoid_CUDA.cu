#include "hip/hip_runtime.h"

#include "hiprand/hiprand_kernel.h"
//#include "helper_cuda.h"
//#include "stdafx.h"
//#include "thrust\sort.h"
//#include "thrust\execution_policy.h"
//#include "device_functions.h"
//#include "math_functions.h"
//#include "thrust 1.8.2\sort.h"


extern "C" void
forCUDA_ShowInfo(const char * info);

extern "C" void
forCUDA_SaveMatAsImage(const char * info,	// input: output path
					   const double * mat,	// input: the mat
					   int w, int h,		// input: the width and height of the mat
					   double valmin,		// input: the minimum value of the mat
					   double valmax		// input: the maximum value of the mat
					   );


//namespace DeepVoid_CUDA
//{

// ���� double �͵ı���������ֵ
__host__ __device__ void
Swap_double(double & A, double & B)
{
	double temp = B;
	B = A;
	A = temp;
}

// ���� int �͵ı���������ֵ
__host__ __device__ void
Swap_int(int & A, int & B)
{
	int temp = B;
	B = A;
	A = temp;
}

// 20161207, computation of matching cost based on NCC (normalized cross-correlation)
// return 0<=val<=2, or -1 if either a or b or both are flat (all array elements are the same)
// ncc(a,b)=ncc(a,kb) (k>0), ncc(a,b)=-ncc(a,kb) (k<0), ncc(a,b)=ncc(a,k+b)
__host__ __device__ double
matchingcost_ncc(const double * a, const double * b, int n)
{
	double sum_a = 0;
	double sum_b = 0;

	for (int i=0; i<n; ++i)
	{
		sum_a += a[i];
		sum_b += b[i];
	}

	double inv_n = 1.0 / n;

	double mean_a = sum_a * inv_n;
	double mean_b = sum_b * inv_n;

	double sum_ai_a_bi_b = 0;
	double sum_ai_a_ai_a = 0;
	double sum_bi_b_bi_b = 0;

	for (int i=0; i<n; ++i)
	{
		double ai_a = a[i] - mean_a;
		double bi_b = b[i] - mean_b;

		sum_ai_a_bi_b += ai_a*bi_b; // sum(ai-a)(bi-b)
		sum_ai_a_ai_a += ai_a*ai_a; // sum(ai-a)(ai-a), the variance or standard deviation of array a
		sum_bi_b_bi_b += bi_b*bi_b; // sum(bi-b)(bi-b), the variance or standard deviation of array b
	}

	double sum_ab = sum_ai_a_ai_a * sum_bi_b_bi_b;

	double rsqrt_sum_ab = rsqrt(sum_ab);

	if (isinf(rsqrt_sum_ab))
	{
		// sum_ab is zero, which means either array a or b or both are flat (all elements are the same)
		return -1;
	}
	else
	{
		double ncc = 1 - sum_ai_a_bi_b*rsqrt_sum_ab; // 0<=val<=2

		return ncc;
	}
}

// �� 1 ά�������ð�ݷ����򣬿�����������Ҳ�ɽ������У������ԭ�����Ԫ����������������������е�����
__host__ __device__ void
Sort_Bubble(double * p,					// ��������������������飬����ִ����������е�Ԫ�ؽ���������߽�������
			int n,						// ���룺���鳤�ȣ�Ԫ�ظ���
			int * idx,					// �����������ɺ��Ԫ��ԭ���������ŵ���������������� 0 ��ʼ
			bool bIsAscent /*= TRUE*/	// ���룺�����򣬻��ǽ�������
			)
{
	int m , k, j, i;
	for( i=0; i<n; i++ )			//Ԥ������������
	{
		idx[i]=i;       
	}

	if(bIsAscent)
	{
		k = 0;
		m = n-1;
		while ( k < m)// k>��m˵���Ѿ��ź�
		{ 	
			j = m-1; 
			m = 0;
			for ( i = k; i <= j; i++)
			{				
				if (p[i] > p[i+1])
				{ 
					Swap_double(p[i], p[i+1]);        // ����������ֵ
					Swap_int(idx[i], idx[i+1]);    // �������������е�����
					m = i;// m�������������
				}
			}
			j = k + 1;
			k = 0;
			for (i = m; i >= j; i--)
			{
				if (p[i-1] > p[i])
				{ 
					Swap_double(p[i-1], p[i]);
					Swap_int(idx[i], idx[i-1]);
					k = i;// k��ǰ������������
				}
			}
		}
	}
	else
	{
		k = 0;
		m = n-1;
		while ( k < m)
		{ 	
			j = m-1; 
			m = 0;
			for ( i = k; i <= j; i++)
			{
				if (p[i] < p[i+1])
				{ 
					Swap_double(p[i], p[i+1]);
					Swap_int(idx[i], idx[i+1]);
					m = i; 
				}
			}
			j = k + 1;
			k = 0;
			for (i = m; i >= j; i--)
			{
				if (p[i-1] < p[i])
				{ 
					Swap_double(p[i-1], p[i]);
					Swap_int(idx[i], idx[i-1]); 
					k = i;
				}
			}
		}
	}
}

// compute the matching cost, proposed by S. Birchfield and C. Tomasi, for a single pixel.
// disparity is along x-axis
__host__ __device__ double 
ComputeMatchingCostforOnePixel_x_BT(int x, int y,				// the coordinates in base image of the checked pixel
									int d,						// the disparity to be checked
									int w, int h,				// the width and height of stereo images
									const unsigned char * imgb,	// input: h*w, the rectified grayscale base image
									const unsigned char * imgm	// input: h*w, the rectified grayscale matching image
									) 
{
	// if the point to be checked xm = x-d is out of border of the matching image,
	// the matching cost is designated with an invalid number, e.g. negative number
	// if d>0, then xm is on the left of x, otherwise, on the right of x
	int xm = x-d;

	if (xm <= 0 || xm >= (w-1) || x <= 0 || x >= (w-1))
	{
		return -1;
	}

	//////////////////////////////////////////////////////////////////////////
	int yw = y * w;

	double fm   = (double)imgm[yw + xm];
	double fm_l = (double)imgm[yw + xm - 1];
	double fm_r = (double)imgm[yw + xm + 1];

	double fm_l_2 = (fm + fm_l) * 0.5;
	double fm_r_2 = (fm + fm_r) * 0.5;

	double pfm[3]; int idx[3];
	pfm[0] = fm; pfm[1] = fm_l_2; pfm[2] = fm_r_2;
//	thrust::sort(thrust::device, pfm, pfm + 3);
	Sort_Bubble(pfm, 3, idx, true);
	double fmmin = pfm[0];
	double fmmax = pfm[2];
	//////////////////////////////////////////////////////////////////////////

	//////////////////////////////////////////////////////////////////////////
	double fb   = double(imgb[yw + x]);
	double fb_l = double(imgb[yw + x - 1]);
	double fb_r = double(imgb[yw + x + 1]);

	double fb_l_2 = (fb + fb_l) * 0.5;
	double fb_r_2 = (fb + fb_r) * 0.5;

	double pfb[3];
	pfb[0] = fb; pfb[1] = fb_l_2; pfb[2] = fb_r_2;
//	thrust::sort(thrust::device, pfb, pfb + 3);
	Sort_Bubble(pfb, 3, idx, true);
	double fbmin = pfb[0];
	double fbmax = pfb[2];
	//////////////////////////////////////////////////////////////////////////

	// in this way
	pfm[0]=0; pfm[1]=fb-fmmax; pfm[2]=fmmin-fb;
//	thrust::sort(thrust::device, pfm, pfm + 3);
	Sort_Bubble(pfm, 3, idx, true);
	double ffm = pfm[2];

	pfb[0]=0; pfb[1]=fm-fbmax; pfb[2]=fbmin-fm;
//	thrust::sort(thrust::device, pfb, pfb + 3);
	Sort_Bubble(pfb, 3, idx, true);
	double ffb = pfb[2];

	if (ffb <= ffm)
	{
		return ffb;
	}

	return ffm;
}

// compute the matching cost, proposed by S. Birchfield and C. Tomasi, for a single pixel.
// disparity is along y-axis
__host__ __device__ double 
ComputeMatchingCostforOnePixel_y_BT(int x, int y,				// input: the coordinates in base image of the checked pixel
									int d,						// input: the disparity to be checked
									int w, int h,				// input: the width and height of stereo images
									const unsigned char * imgb,	// input: h*w, the rectified grayscale base image
									const unsigned char * imgm	// input: h*w, the rectified grayscale matching image
									)
{
	// if the point to be checked ym = y-d is out of border of the matching image,
	// the matching cost is designated with an invalid number, e.g. negative number
	// if d>0, then ym is on the top of y, otherwise, on the bottom of y
	int ym = y - d;

	if (ym <= 0 || ym >= (h - 1) || y <= 0 || y >= (h - 1))
	{
		return -1;
	}

	//////////////////////////////////////////////////////////////////////////
	int ymw = ym*w;

	double fm   = (double)imgm[ymw + x];
	double fm_l = (double)imgm[ymw - w + x];
	double fm_r = (double)imgm[ymw + w + x];

	double fm_l_2 = (fm + fm_l) * 0.5;
	double fm_r_2 = (fm + fm_r) * 0.5;

	double pfm[3]; int idx[3];
	pfm[0] = fm; pfm[1] = fm_l_2; pfm[2] = fm_r_2;
//	thrust::sort(/*thrust::device,*/ pfm, pfm + 3);
	Sort_Bubble(pfm, 3, idx, true);
	double fmmin = pfm[0];
	double fmmax = pfm[2];
	//////////////////////////////////////////////////////////////////////////

	//////////////////////////////////////////////////////////////////////////
	int yw = y*w;

	double fb   = double(imgb[yw + x]);
	double fb_l = double(imgb[yw - w + x]);
	double fb_r = double(imgb[yw + w + x]);

	double fb_l_2 = (fb + fb_l) * 0.5;
	double fb_r_2 = (fb + fb_r) * 0.5;

	double pfb[3];
	pfb[0] = fb; pfb[1] = fb_l_2; pfb[2] = fb_r_2;
//	thrust::sort(/*thrust::device,*/ pfb, pfb + 3);
	Sort_Bubble(pfb, 3, idx, true);
	double fbmin = pfb[0];
	double fbmax = pfb[2];
	//////////////////////////////////////////////////////////////////////////

	// in this way
	pfm[0] = 0; pfm[1] = fb - fmmax; pfm[2] = fmmin - fb;
//	thrust::sort(/*thrust::device,*/ pfm, pfm + 3);
	Sort_Bubble(pfm, 3, idx, true);
	double ffm = pfm[2]; // got to be non-negative

	pfb[0] = 0; pfb[1] = fm - fbmax; pfb[2] = fbmin - fm;
//	thrust::sort(/*thrust::device, */pfb, pfb + 3);
	Sort_Bubble(pfb, 3, idx, true);
	double ffb = pfb[2]; // got to be non-negative

	if (ffb <= ffm)
	{
		return ffb;
	}

	return ffm;
}

// compute the matching cost based on normalized cross-correlation (ncc).
// disparity is along x-axis
__host__ __device__ double 
ComputeMatchingCostforOnePixel_x_ncc(int x, int y,				// the coordinates in base image of the checked pixel
									 int d,						// the disparity to be checked
									 int w, int h,				// the width and height of stereo images
									 int hw, int hh,			// the half width and half height of the support window
									 const unsigned char * imgb,// input: h*w, the rectified grayscale base image
									 const unsigned char * imgm	// input: h*w, the rectified grayscale matching image
									 ) 
{
	// if the point to be checked xm = x-d is out of border of the matching image,
	// the matching cost is designated with an invalid number, e.g. negative number
	// if d>0, then xm is on the left of x, otherwise, on the right of x
	int xm = x-d;

	if ((y-hh) < 0 || (y+hh) >= h || (x-hw) < 0 || (x+hw) >= w || (xm-hw) < 0 || (xm+hw) >= w)
	{
		return -1;
	}

	int n = (2 * hh + 1)*(2 * hw + 1);
	double inv_n = 1.0 / n;

	//////////////////////////////////////////////////////////////////////////
	// compute the means of each support windows
	double sum_b = 0;
	double sum_m = 0;

	for (int i = -hh; i <= hh; ++i)
	{
		for (int j = -hw; j <= hw; ++j)
		{
			sum_b += imgb[(y + i)*w + (x + j)];
			sum_m += imgm[(y + i)*w + (xm + j)];
		}
	}

	double mean_b = sum_b*inv_n;
	double mean_m = sum_m*inv_n;
	//////////////////////////////////////////////////////////////////////////

	//////////////////////////////////////////////////////////////////////////
	// compute the standard deviation and final ncc value
	double sum_bijmij = 0;
	double sum_bijbij = 0;
	double sum_mijmij = 0;

	for (int i = -hh; i <= hh; ++i)
	{
		for (int j = -hw; j <= hw; ++j)
		{
			double bij = imgb[(y + i)*w + (x + j)] - mean_b;
			double mij = imgm[(y + i)*w + (xm + j)] - mean_m;

			sum_bijmij += bij*mij;
			sum_bijbij += bij*bij;
			sum_mijmij += mij*mij;
		}
	}

	double bbmm = sum_bijbij * sum_mijmij;

	double rsqrt_bbmm = rsqrt(bbmm);

	if (isinf(rsqrt_bbmm))
	{
		// sum_ab is zero, which means either array a or b or both are flat (all elements are the same)
		return -1;
	}
	else
	{
		double ncc = 1 - sum_bijmij*rsqrt_bbmm; // 0<=val<=2

		return ncc;
	}
	//////////////////////////////////////////////////////////////////////////
}

// compute the matching cost based on normalized cross-correlation (ncc).
// disparity is along y-axis
__host__ __device__ double 
ComputeMatchingCostforOnePixel_y_ncc(int x, int y,				// input: the coordinates in base image of the checked pixel
									 int d,						// input: the disparity to be checked
									 int w, int h,				// input: the width and height of stereo images
									 int hw, int hh,			// the half width and half height of the support window
									 const unsigned char * imgb,// input: h*w, the rectified grayscale base image
									 const unsigned char * imgm	// input: h*w, the rectified grayscale matching image
									 )
{
	// if the point to be checked ym = y-d is out of border of the matching image,
	// the matching cost is designated with an invalid number, e.g. negative number
	// if d>0, then ym is on the top of y, otherwise, on the bottom of y
	int ym = y - d;

	if ((x - hw) < 0 || (x + hw) >= w || (y - hh) < 0 || (y + hh) >= h || (ym - hh) < 0 || (ym + hh) >= h)
	{
		return -1;
	}

	int n = (2 * hh + 1)*(2 * hw + 1);
	double inv_n = 1.0 / n;

	//////////////////////////////////////////////////////////////////////////
	// compute the means of each support windows
	double sum_b = 0;
	double sum_m = 0;

	for (int i = -hh; i <= hh; ++i)
	{
		for (int j = -hw; j <= hw; ++j)
		{
			sum_b += imgb[(y + i)*w + (x + j)];
			sum_m += imgm[(ym + i)*w + (x + j)];
		}
	}

	double mean_b = sum_b*inv_n;
	double mean_m = sum_m*inv_n;
	//////////////////////////////////////////////////////////////////////////

	//////////////////////////////////////////////////////////////////////////
	// compute the standard deviation and final ncc value
	double sum_bijmij = 0;
	double sum_bijbij = 0;
	double sum_mijmij = 0;

	for (int i = -hh; i <= hh; ++i)
	{
		for (int j = -hw; j <= hw; ++j)
		{
			double bij = imgb[(y + i)*w + (x + j)] - mean_b;
			double mij = imgm[(ym + i)*w + (x + j)] - mean_m;

			sum_bijmij += bij*mij;
			sum_bijbij += bij*bij;
			sum_mijmij += mij*mij;
		}
	}

	double bbmm = sum_bijbij * sum_mijmij;

	double rsqrt_bbmm = rsqrt(bbmm);

	if (isinf(rsqrt_bbmm))
	{
		// sum_ab is zero, which means either array a or b or both are flat (all elements are the same)
		return -1;
	}
	else
	{
		double ncc = 1 - sum_bijmij*rsqrt_bbmm; // 0<=val<=2

		return ncc;
	}
	//////////////////////////////////////////////////////////////////////////
}

// compute the matching cost based on normalized cross-correlation (ncc).
// disparity is along x-axis
__host__ __device__ double 
ComputeMatchingCostforOnePixel_x_opencvncc(int x, int y,				// the coordinates in base image of the checked pixel
										   int d,						// the disparity to be checked
										   int w, int h,				// the width and height of stereo images
										   int hw, int hh,				// the half width and half height of the support window
										   const unsigned char * imgb,	// input: h*w, the rectified grayscale base image
										   const unsigned char * imgm	// input: h*w, the rectified grayscale matching image
										   )
{
	// if the point to be checked xm = x-d is out of border of the matching image,
	// the matching cost is designated with an invalid number, e.g. negative number
	// if d>0, then xm is on the left of x, otherwise, on the right of x
	int xm = x-d;

	if ((y-hh) < 0 || (y+hh) >= h || (x-hw) < 0 || (x+hw) >= w || (xm-hw) < 0 || (xm+hw) >= w)
	{
		return -1;
	}

	//////////////////////////////////////////////////////////////////////////
	// compute the standard deviation and final ncc value
	double sum_bijmij = 0;
	double sum_bijbij = 0;
	double sum_mijmij = 0;

	for (int i = -hh; i <= hh; ++i)
	{
		for (int j = -hw; j <= hw; ++j)
		{
			double bij = imgb[(y + i)*w + (x + j)];
			double mij = imgm[(y + i)*w + (xm + j)];

			sum_bijmij += bij*mij;
			sum_bijbij += bij*bij;
			sum_mijmij += mij*mij;
		}
	}

	double bbmm = sum_bijbij * sum_mijmij;

	double rsqrt_bbmm = rsqrt(bbmm);

	if (isinf(rsqrt_bbmm))
	{
		// bbmm is zero, which means either array a or b or both are all zeros
		return -1;
	}
	else
	{
		double ncc = 1 - sum_bijmij*rsqrt_bbmm; // 0<=val<=1

		return ncc;
	}
	//////////////////////////////////////////////////////////////////////////
}

// compute the matching cost based on normalized cross-correlation (ncc).
// disparity is along y-axis
__host__ __device__ double 
ComputeMatchingCostforOnePixel_y_opencvncc(int x, int y,				// input: the coordinates in base image of the checked pixel
										   int d,						// input: the disparity to be checked
										   int w, int h,				// input: the width and height of stereo images
										   int hw, int hh,				// the half width and half height of the support window
										   const unsigned char * imgb,	// input: h*w, the rectified grayscale base image
										   const unsigned char * imgm	// input: h*w, the rectified grayscale matching image
										   )
{
	// if the point to be checked ym = y-d is out of border of the matching image,
	// the matching cost is designated with an invalid number, e.g. negative number
	// if d>0, then ym is on the top of y, otherwise, on the bottom of y
	int ym = y - d;

	if ((x - hw) < 0 || (x + hw) >= w || (y - hh) < 0 || (y + hh) >= h || (ym - hh) < 0 || (ym + hh) >= h)
	{
		return -1;
	}

	//////////////////////////////////////////////////////////////////////////
	// compute the standard deviation and final ncc value
	double sum_bijmij = 0;
	double sum_bijbij = 0;
	double sum_mijmij = 0;

	for (int i = -hh; i <= hh; ++i)
	{
		for (int j = -hw; j <= hw; ++j)
		{
			double bij = imgb[(y + i)*w + (x + j)];
			double mij = imgm[(ym + i)*w + (x + j)];

			sum_bijmij += bij*mij;
			sum_bijbij += bij*bij;
			sum_mijmij += mij*mij;
		}
	}

	double bbmm = sum_bijbij * sum_mijmij;

	double rsqrt_bbmm = rsqrt(bbmm);

	if (isinf(rsqrt_bbmm))
	{
		// bbmm is zero, which means either array a or b or both are all zeros
		return -1;
	}
	else
	{
		double ncc = 1 - sum_bijmij*rsqrt_bbmm; // 0<=val<=1

		return ncc;
	}
	//////////////////////////////////////////////////////////////////////////
}

// compute the matching cost based on normalized cross-correlation (ncc).
// disparity is along y-axis
__host__ __device__ void 
previous_ij(int i, int j,				// input: image indices of current pixel
			int & i_pre, int & j_pre,	// output:image indices of previous pixel
	        int dir						// input: 0:(0,-1), 1:(1,-1), 2:(1,0), 3:(1,1), 4:(0,1), 5:(-1,1), 6:(-1,0), 7:(-1,-1)
			)
{
	i_pre = i;
	j_pre = j;

	if (0 == dir)
	{
		--i_pre;
	}
	else if (1 == dir)
	{
		--i_pre;
		++j_pre;
	}
	else if (2 == dir)
	{
		++j_pre;
	}
	else if (3 == dir)
	{
		++i_pre;
		++j_pre;
	}
	else if (4 == dir)
	{
		++i_pre;
	}
	else if (5 == dir)
	{
		++i_pre;
		--j_pre;
	}
	else if (6 == dir)
	{
		--j_pre;
	}
	else if (7 == dir)
	{
		--i_pre;
		--j_pre;
	}
	else
	{

	}
}

// compute the matching cost based on normalized cross-correlation (ncc).
// disparity is along y-axis
__host__ __device__ void 
CostAggregation_OnePixel(int w, int h, int nd,	// input: the width and height of stereo images
						 double * C,			// output:h*w*nd, nd = (dmax - dmin + 1), the output Disparity Space Image
						 int i, int j,			// input: image indices of current pixel
						 int i_pre, int j_pre,	// input: image indices of previous pixel
						 double P1,				// input: constant penalty pixels in the neigborhood of (x,y), for which the disparity changes a little bit (that is 1 pixel)
						 double P2				// input: a larger constant penalty for all larger disparity changes
						 )
{
	if (i<0 || i>=h || j<0 || j>=w || i_pre<0 || i_pre>= h || j_pre<0 || j_pre>=w) // cross the boundary
	{
		return;
	}

	int nPix = w*h; // number of image pixels
	int wi_j_cur = w*i + j;
	int wi_j_pre = w*i_pre + j_pre;

	for (int k = 0; k < nd; ++k)
	{
		double val_cur_k = C[nPix*k + wi_j_cur]; // matching cost of current pixel at disparity level k

		if (val_cur_k < 0)
		{
			continue;
		}
		
		bool bAllNeg = true; // all aggregated costs of previous pixel are negative

		double minPre;  // record the min nonnegative value of aggregated costs of previous pixel
		double minCost; // record the min cost aggregated from all valid disparity levels of previous pixel

		for (int k1 = 0; k1 < nd; ++k1)
		{
			double val_pre_k1 = C[nPix*k1 + wi_j_pre]; // aggregated cost of previous pixel at disparity level k1

			if (val_pre_k1 < 0)
			{
				continue;
			}

			double curCost = val_pre_k1;

			if (k == k1) // the same disparity level
			{
				// do nothing
			}
			else if (abs(k - k1) == 1) // the disparity difference is 1 pixel
			{
				curCost += P1;
			}
			else // the disparity difference is 2 or more pixels
			{
				curCost += P2;
			}

			if (bAllNeg)
			{
				// arriving here means that the first non-negative aggregated cost value of previous pixel is found
				minPre = val_pre_k1;
				minCost = curCost;
				bAllNeg = false;
			}
			else
			{
				if (val_pre_k1 < minPre)
				{
					minPre = val_pre_k1;
				}
				
				if (curCost < minCost)
				{
					minCost = curCost;
				}
			}
		}

		if (bAllNeg) // break outta the loop if all aggregated costs of previous pixel are really negative
		{
			break;
		}

		C[nPix*k + wi_j_cur] += (minCost - minPre);
	}
}

// 20161226, new DSI layout
__host__ __device__ void 
CostAggregation_OnePixel_new(int w, int h, int nd,	// input: the width and height of stereo images
							 double * C,			// output:h*w*nd, nd = (dmax - dmin + 1), the output Disparity Space Image
							 int i, int j,			// input: image indices of current pixel
							 int i_pre, int j_pre,	// input: image indices of previous pixel
							 double P1,				// input: constant penalty pixels in the neigborhood of (x,y), for which the disparity changes a little bit (that is 1 pixel)
							 double P2				// input: a larger constant penalty for all larger disparity changes
							 )
{
	int offset_cur = (w*i + j)*nd;
	int offset_pre = (w*i_pre + j_pre)*nd;

	for (int k = 0; k < nd; ++k)
	{
		double & val_cur_k = C[offset_cur + k]; // matching cost of current pixel at disparity level k

		if (val_cur_k < 0)
		{
			continue;
		}
		
		bool bAllNeg = true; // all aggregated costs of previous pixel are negative

		double minPre;  // record the min nonnegative value of aggregated costs of previous pixel
		double minCost; // record the min cost aggregated from all valid disparity levels of previous pixel

		for (int k1 = 0; k1 < nd; ++k1)
		{
			double val_pre_k1 = C[offset_pre + k1]; // aggregated cost of previous pixel at disparity level k1

			if (val_pre_k1 < 0)
			{
				continue;
			}

			double curCost = val_pre_k1;

			if (k == k1) // the same disparity level
			{
				// do nothing
			}
			else if (abs(k - k1) == 1) // the disparity difference is 1 pixel
			{
				curCost += P1;
			}
			else // the disparity difference is 2 or more pixels
			{
				curCost += P2;
			}

			if (bAllNeg)
			{
				// arriving here means that the first non-negative aggregated cost value of previous pixel is found
				minPre = val_pre_k1;
				minCost = curCost;
				bAllNeg = false;
			}
			else
			{
				if (val_pre_k1 < minPre)
				{
					minPre = val_pre_k1;
				}
				
				if (curCost < minCost)
				{
					minCost = curCost;
				}
			}
		}

		if (bAllNeg) // break outta the loop if all aggregated costs of previous pixel are really negative
		{
			break;
		}

		val_cur_k += (minCost - minPre);
	}
}

// 20161228, new DSI layout
__host__ __device__ void 
CostAggregation_OnePixel_new(int w, int h, int nd,	// input: the width and height of stereo images
							 double * C,			// output:h*w*nd, nd = (dmax - dmin + 1), the output Disparity Space Image
							 int i, int j, int k,	// input: image indices of current pixel, and index of current disparity level
							 int i_pre, int j_pre,	// input: image indices of previous pixel
							 double P1,				// input: constant penalty pixels in the neigborhood of (x,y), for which the disparity changes a little bit (that is 1 pixel)
							 double P2				// input: a larger constant penalty for all larger disparity changes
							 )
{
	int offset_cur = (w*i + j)*nd;
	int offset_pre = (w*i_pre + j_pre)*nd;

	double & val_cur_k = C[offset_cur + k]; // matching cost of current pixel at disparity level k

	if (val_cur_k < 0)
	{
		return;
	}
		
	bool bAllNeg = true; // all aggregated costs of previous pixel are negative

	double minPre;  // record the min nonnegative value of aggregated costs of previous pixel
	double minCost; // record the min cost aggregated from all valid disparity levels of previous pixel

	for (int k1 = 0; k1 < nd; ++k1)
	{
		double val_pre_k1 = C[offset_pre + k1]; // aggregated cost of previous pixel at disparity level k1

		if (val_pre_k1 < 0)
		{
			continue;
		}

		double curCost = val_pre_k1;

		if (k == k1) // the same disparity level
		{
			// do nothing
		}
		else if (abs(k - k1) == 1) // the disparity difference is 1 pixel
		{
			curCost += P1;
		}
		else // the disparity difference is 2 or more pixels
		{
			curCost += P2;
		}

		if (bAllNeg)
		{
			// arriving here means that the first non-negative aggregated cost value of previous pixel is found
			minPre = val_pre_k1;
			minCost = curCost;
			bAllNeg = false;
		}
		else
		{
			if (val_pre_k1 < minPre)
			{
				minPre = val_pre_k1;
			}
				
			if (curCost < minCost)
			{
				minCost = curCost;
			}
		}
	}

	if (bAllNeg) // break outta the loop if all aggregated costs of previous pixel are really negative
	{
		return;
	}

	val_cur_k += (minCost - minPre);
}

__host__ __device__ void 
CostAggregation_OnePixel(int w, int h, int nd,	// input: the width and height of stereo images
						 int threadIdx,			// input: the index of current thread within the thread block
						 const double * s_pre,	// input: shared memory of aggregated costs of all previous pixels within the same thread block
						 double * C,			// input & output:h*w*nd, nd = (dmax - dmin + 1), the output Disparity Space Image
						 int i, int j,			// input: image indices of current pixel
						 double P1,				// input: constant penalty pixels in the neigborhood of (x,y), for which the disparity changes a little bit (that is 1 pixel)
						 double P2				// input: a larger constant penalty for all larger disparity changes
						 )
{
	if (i<0 || i>=h || j<0 || j>=w) // cross the boundary
	{
		return;
	}

	int nPix = w*h; // number of image pixels
	int wi_j_cur = w*i + j;
	int tidxnd = threadIdx*nd;

	for (int k = 0; k < nd; ++k)
	{
		double & val_cur_k = C[nPix*k + wi_j_cur]; // matching cost of current pixel at disparity level k

		if (val_cur_k < 0)
		{
			continue;
		}
		
		bool bAllNeg = true; // all aggregated costs of previous pixel are negative

		double minPre;  // record the min nonnegative value of aggregated costs of previous pixel
		double minCost; // record the min cost aggregated from all valid disparity levels of previous pixel

		for (int k1 = 0; k1 < nd; ++k1)
		{
			double val_pre_k1 = s_pre[tidxnd + k1]; // aggregated cost of previous pixel at disparity level k1

			if (val_pre_k1 < 0)
			{
				continue;
			}

			double curCost = val_pre_k1;

			if (k == k1) // the same disparity level
			{
				// do nothing
			}
			else if (abs(k - k1) == 1) // the disparity difference is 1 pixel
			{
				curCost += P1;
			}
			else // the disparity difference is 2 or more pixels
			{
				curCost += P2;
			}

			if (bAllNeg)
			{
				// arriving here means that the first non-negative aggregated cost value of previous pixel is found
				minPre = val_pre_k1;
				minCost = curCost;
				bAllNeg = false;
			}
			else
			{
				if (val_pre_k1 < minPre)
				{
					minPre = val_pre_k1;
				}
				
				if (curCost < minCost)
				{
					minCost = curCost;
				}
			}
		}

		if (bAllNeg) // break outta the loop if all aggregated costs of previous pixel are really negative
		{
			break;
		}

		val_cur_k += (minCost - minPre);
	}
}

// compute the matching cost based on normalized cross-correlation (ncc).
// disparity is along y-axis
__host__ __device__ void 
CostAggregation_OnePixel(double * cur,			// input & output: it is DSI values as input, aggregated costs as output
						 const double * pre,	// input: it is aggregated costs of the previous pixel
						 int n,					// input: number of disparity levels
						 double P1,				// input: constant penalty pixels in the neigborhood of (x,y), for which the disparity changes a little bit (that is 1 pixel)
						 double P2				// input: a larger constant penalty for all larger disparity changes
						 )
{
	for (int k = 0; k < n; ++k)
	{
		double & val_cur_k = cur[k]; // matching cost of current pixel at disparity level k

		if (val_cur_k < 0)
		{
			continue;
		}
		
		bool bAllNeg = true; // all aggregated costs of previous pixel are negative

		double minPre;  // record the min nonnegative value of aggregated costs of previous pixel
		double minCost; // record the min cost aggregated from all valid disparity levels of previous pixel

		for (int k1 = 0; k1 < n; ++k1)
		{
			double val_pre_k1 = pre[k1]; // aggregated cost of previous pixel at disparity level k1

			if (val_pre_k1 < 0)
			{
				continue;
			}

			double curCost = val_pre_k1;

			if (k == k1) // the same disparity level
			{
				// do nothing
			}
			else if (abs(k - k1) == 1) // the disparity difference is 1 pixel
			{
				curCost += P1;
			}
			else // the disparity difference is 2 or more pixels
			{
				curCost += P2;
			}

			if (bAllNeg)
			{
				// arriving here means that the first non-negative aggregated cost value of previous pixel is found
				minPre = val_pre_k1;
				minCost = curCost;
				bAllNeg = false;
			}
			else
			{
				if (val_pre_k1 < minPre)
				{
					minPre = val_pre_k1;
				}
				
				if (curCost < minCost)
				{
					minCost = curCost;
				}
			}
		}

		if (bAllNeg) // break outta the loop if all aggregated costs of previous pixel are really negative
		{
			break;
		}

		val_cur_k += (minCost - minPre);
	}
}

// 20161115, four directions: 0:left to right. 1:right to left. 2:up to bottom. 3:bottom to up
// Generate the Disparity Space Image (DSI) based on the pixelwise matching
// cost proposed in <Depth Discontinuities by Pixel-to-Pixel Stereo> by S. Birchfield and C. Tomasi
__global__ void 
kernel_GenerateDSI_l2r_BT(int w, int h,					// input: the width and height of stereo images
						  const unsigned char * imgb,	// input: h*w, the rectified grayscale base image
						  const unsigned char * imgm,	// input: h*w, the rectified grayscale matching image
						  int dmin,						// input: scalar, the minimal disparity, can be negative, but make sure dmin<dmax
						  int dmax,						// input: scalar, the maximal disparity, can be negative, but make sure dmin<dmax
						  double * DSI					// output:h*w*nd, nd = (dmax - dmin + 1), the output Disparity Space Image
						  )
{
	// the pixel indices
	int i = blockDim.y * blockIdx.y + threadIdx.y;
	int j = blockDim.x * blockIdx.x + threadIdx.x;

	if (i >= h || j >= w) // over the boundary
	{
		return;
	}

	int nd = dmax - dmin + 1; // number of disparities
	int wh = w*h; // number of pixels in the image
	int iwj = i*w + j;
	
	for (int d = dmin; d <= dmax; ++d)
	{
		int k = d - dmin; // vertical index

		DSI[k*wh + iwj] = ComputeMatchingCostforOnePixel_x_BT(j, i, d, w, h, imgb, imgm); // DSI[k*w*h + i*w + j]
	}
}

// 20161115, four directions: 0:left to right. 1:right to left. 2:up to bottom. 3:bottom to up
// Generate the Disparity Space Image (DSI) based on the pixelwise matching
// cost proposed in <Depth Discontinuities by Pixel-to-Pixel Stereo> by S. Birchfield and C. Tomasi
__global__ void 
kernel_GenerateDSI_r2l_BT(int w, int h,					// input: the width and height of stereo images
						  const unsigned char * imgb,	// input: h*w, the rectified grayscale base image
						  const unsigned char * imgm,	// input: h*w, the rectified grayscale matching image
						  int dmin,						// input: scalar, the minimal disparity, can be negative, but make sure dmin<dmax
						  int dmax,						// input: scalar, the maximal disparity, can be negative, but make sure dmin<dmax
						  double * DSI					// output:h*w*nd, nd = (dmax - dmin + 1), the output Disparity Space Image
						  )
{
	// the pixel indices
	int i = blockDim.y * blockIdx.y + threadIdx.y;
	int j = blockDim.x * blockIdx.x + threadIdx.x;

	if (i >= h || j >= w) // over the boundary
	{
		return;
	}

	int nd = dmax - dmin + 1; // number of disparities
	int wh = w*h; // number of pixels in the image
	int iwj = i*w + j;
	
	for (int d = dmin; d <= dmax; ++d)
	{
		int k = d - dmin; // vertical index

		DSI[k*wh + iwj] = ComputeMatchingCostforOnePixel_x_BT(j, i, -d, w, h, imgm, imgb); // DSI[k*w*h + i*w + j]
	}
}

// 20161115, four directions: 0:left to right. 1:right to left. 2:up to bottom. 3:bottom to up
// Generate the Disparity Space Image (DSI) based on the pixelwise matching
// cost proposed in <Depth Discontinuities by Pixel-to-Pixel Stereo> by S. Birchfield and C. Tomasi
__global__ void 
kernel_GenerateDSI_u2b_BT(int w, int h,					// input: the width and height of stereo images
						  const unsigned char * imgb,	// input: h*w, the rectified grayscale base image
						  const unsigned char * imgm,	// input: h*w, the rectified grayscale matching image
						  int dmin,						// input: scalar, the minimal disparity, can be negative, but make sure dmin<dmax
						  int dmax,						// input: scalar, the maximal disparity, can be negative, but make sure dmin<dmax
						  double * DSI					// output:h*w*nd, nd = (dmax - dmin + 1), the output Disparity Space Image
						  )
{
	// the pixel indices
	int i = blockDim.y * blockIdx.y + threadIdx.y;
	int j = blockDim.x * blockIdx.x + threadIdx.x;

	if (i >= h || j >= w) // over the boundary
	{
		return;
	}

	int nd = dmax - dmin + 1; // number of disparities
	int wh = w*h; // number of pixels in the image
	int iwj = i*w + j;
	
	for (int d = dmin; d <= dmax; ++d)
	{
		int k = d - dmin; // vertical index

		DSI[k*wh + iwj] = ComputeMatchingCostforOnePixel_y_BT(j, i, d, w, h, imgb, imgm); // DSI[k*w*h + i*w + j]
	}
}

// 20161115, four directions: 0:left to right. 1:right to left. 2:up to bottom. 3:bottom to up
// Generate the Disparity Space Image (DSI) based on the pixelwise matching
// cost proposed in <Depth Discontinuities by Pixel-to-Pixel Stereo> by S. Birchfield and C. Tomasi
__global__ void 
kernel_GenerateDSI_b2u_BT(int w, int h,					// input: the width and height of stereo images
						  const unsigned char * imgb,	// input: h*w, the rectified grayscale base image
						  const unsigned char * imgm,	// input: h*w, the rectified grayscale matching image
						  int dmin,						// input: scalar, the minimal disparity, can be negative, but make sure dmin<dmax
						  int dmax,						// input: scalar, the maximal disparity, can be negative, but make sure dmin<dmax
						  double * DSI					// output:h*w*nd, nd = (dmax - dmin + 1), the output Disparity Space Image
						  )
{
	// the pixel indices
	int i = blockDim.y * blockIdx.y + threadIdx.y;
	int j = blockDim.x * blockIdx.x + threadIdx.x;

	if (i >= h || j >= w) // over the boundary
	{
		return;
	}

	int nd = dmax - dmin + 1; // number of disparities
	int wh = w*h; // number of pixels in the image
	int iwj = i*w + j;
	
	for (int d = dmin; d <= dmax; ++d)
	{
		int k = d - dmin; // vertical index

		DSI[k*wh + iwj] = ComputeMatchingCostforOnePixel_y_BT(j, i, -d, w, h, imgm, imgb); // DSI[k*w*h + i*w + j]
	}
}

// 20161115, four directions: 0:left to right. 1:right to left. 2:up to bottom. 3:bottom to up
// Generate the Disparity Space Image (DSI) based on the pixelwise matching
// cost proposed in <Depth Discontinuities by Pixel-to-Pixel Stereo> by S. Birchfield and C. Tomasi
__global__ void 
kernel_GenerateDSI(int w, int h,				// input: the width and height of stereo images
				   int hw, int hh,				// the half width and half height of the support window
				   const unsigned char * imgb,	// input: h*w, the rectified grayscale base image
				   const unsigned char * imgm,	// input: h*w, the rectified grayscale matching image
				   int dmin,					// input: scalar, the minimal disparity, can be negative, but make sure dmin<dmax
				   int dmax,					// input: scalar, the maximal disparity, can be negative, but make sure dmin<dmax
				   double * DSI,				// output:h*w*nd, nd = (dmax - dmin + 1), the output Disparity Space Image
				   int dir,						// input: 0:l2r, 1:r2l, 2:u2b, 3:b2u
				   int costType                 // input: 0:BT,  1:ncc
				   )
{
	// the pixel indices
	int i = blockDim.y * blockIdx.y + threadIdx.y;
	int j = blockDim.x * blockIdx.x + threadIdx.x;

	if (i >= h || j >= w) // over the boundary
	{
		return;
	}

	int nd = dmax - dmin + 1; // number of disparities
	int wh = w*h; // number of pixels in the image
	int iwj = i*w + j;

	if (0==dir) // l2r
	{
		if (0==costType) // BT
		{
			for (int d = dmin; d <= dmax; ++d)
			{
				int k = d - dmin; // vertical index

				DSI[k*wh + iwj] = ComputeMatchingCostforOnePixel_x_BT(j, i, d, w, h, imgb, imgm); // DSI[k*w*h + i*w + j]
			}
		}
		else if (1==costType) // ncc
		{
			for (int d = dmin; d <= dmax; ++d)
			{
				int k = d - dmin; // vertical index

				DSI[k*wh + iwj] = ComputeMatchingCostforOnePixel_x_ncc(j, i, d, w, h, hw, hh, imgb, imgm); // DSI[k*w*h + i*w + j]
			}
		}
		else if (2 == costType) // opencv ncc
		{
			for (int d = dmin; d <= dmax; ++d)
			{
				int k = d - dmin; // vertical index

				DSI[k*wh + iwj] = ComputeMatchingCostforOnePixel_x_opencvncc(j, i, d, w, h, hw, hh, imgb, imgm); // DSI[k*w*h + i*w + j]
			}
		}
		else
		{

		}
	}
	else if (1==dir) // r2l
	{
		if (0 == costType) // BT
		{
			for (int d = dmin; d <= dmax; ++d)
			{
				int k = d - dmin; // vertical index

				DSI[k*wh + iwj] = ComputeMatchingCostforOnePixel_x_BT(j, i, -d, w, h, imgm, imgb); // DSI[k*w*h + i*w + j]
			}
		}
		else if (1 == costType) // ncc
		{
			for (int d = dmin; d <= dmax; ++d)
			{
				int k = d - dmin; // vertical index

				DSI[k*wh + iwj] = ComputeMatchingCostforOnePixel_x_ncc(j, i, -d, w, h, hw, hh, imgm, imgb); // DSI[k*w*h + i*w + j]
			}
		}
		else if (2 == costType) // opencv ncc
		{
			for (int d = dmin; d <= dmax; ++d)
			{
				int k = d - dmin; // vertical index

				DSI[k*wh + iwj] = ComputeMatchingCostforOnePixel_x_opencvncc(j, i, -d, w, h, hw, hh, imgm, imgb); // DSI[k*w*h + i*w + j]
			}
		}
		else
		{

		}
	}
	else if (2==dir) // u2b
	{
		if (0 == costType) // BT
		{
			for (int d = dmin; d <= dmax; ++d)
			{
				int k = d - dmin; // vertical index

				DSI[k*wh + iwj] = ComputeMatchingCostforOnePixel_y_BT(j, i, d, w, h, imgb, imgm); // DSI[k*w*h + i*w + j]
			}
		}
		else if (1 == costType) // ncc
		{
			for (int d = dmin; d <= dmax; ++d)
			{
				int k = d - dmin; // vertical index

				DSI[k*wh + iwj] = ComputeMatchingCostforOnePixel_y_ncc(j, i, d, w, h, hw, hh, imgb, imgm); // DSI[k*w*h + i*w + j]
			}
		}
		else if (2 == costType) // opencv ncc
		{
			for (int d = dmin; d <= dmax; ++d)
			{
				int k = d - dmin; // vertical index

				DSI[k*wh + iwj] = ComputeMatchingCostforOnePixel_y_opencvncc(j, i, d, w, h, hw, hh, imgb, imgm); // DSI[k*w*h + i*w + j]
			}
		}
		else
		{

		}
	}
	else // b2u
	{
		if (0 == costType) // BT
		{
			for (int d = dmin; d <= dmax; ++d)
			{
				int k = d - dmin; // vertical index

				DSI[k*wh + iwj] = ComputeMatchingCostforOnePixel_y_BT(j, i, -d, w, h, imgm, imgb); // DSI[k*w*h + i*w + j]
			}
		}
		else if (1 == costType) // ncc
		{
			for (int d = dmin; d <= dmax; ++d)
			{
				int k = d - dmin; // vertical index

				DSI[k*wh + iwj] = ComputeMatchingCostforOnePixel_y_ncc(j, i, -d, w, h, hw, hh, imgm, imgb); // DSI[k*w*h + i*w + j]
			}
		}
		else if (2 == costType) // opencv ncc
		{
			for (int d = dmin; d <= dmax; ++d)
			{
				int k = d - dmin; // vertical index

				DSI[k*wh + iwj] = ComputeMatchingCostforOnePixel_y_opencvncc(j, i, -d, w, h, hw, hh, imgm, imgb); // DSI[k*w*h + i*w + j]
			}
		}
		else
		{

		}
	}
}

// 20161226, DSI with new layout
// 20161115, four directions: 0:left to right. 1:right to left. 2:up to bottom. 3:bottom to up
// Generate the Disparity Space Image (DSI) based on the pixelwise matching
// cost proposed in <Depth Discontinuities by Pixel-to-Pixel Stereo> by S. Birchfield and C. Tomasi
__global__ void 
kernel_GenerateDSI_new(int w, int h,				// input: the width and height of stereo images
					   int hw, int hh,				// the half width and half height of the support window
					   const unsigned char * imgb,	// input: h*w, the rectified grayscale base image
					   const unsigned char * imgm,	// input: h*w, the rectified grayscale matching image
					   int dmin,					// input: scalar, the minimal disparity, can be negative, but make sure dmin<dmax
					   int dmax,					// input: scalar, the maximal disparity, can be negative, but make sure dmin<dmax
					   double * DSI,				// output:h*w*nd, nd = (dmax - dmin + 1), the output Disparity Space Image
					   int dir,						// input: 0:l2r, 1:r2l, 2:u2b, 3:b2u
					   int costType                 // input: 0:BT,  1:ncc
					   )
{
	// the pixel indices
	int i = blockDim.y * blockIdx.y + threadIdx.y;
	int j = blockDim.x * blockIdx.x + threadIdx.x;

	if (i >= h || j >= w) // over the boundary
	{
		return;
	}

	int nd = dmax - dmin + 1; // number of disparities
	int offset = (i*w + j)*nd;

	if (0==dir) // l2r
	{
		if (0==costType) // BT
		{
			for (int d = dmin; d <= dmax; ++d)
			{
				int k = d - dmin; // vertical index

				DSI[offset + k] = ComputeMatchingCostforOnePixel_x_BT(j, i, d, w, h, imgb, imgm); // DSI[k*w*h + i*w + j]
			}
		}
		else if (1==costType) // ncc
		{
			for (int d = dmin; d <= dmax; ++d)
			{
				int k = d - dmin; // vertical index

				DSI[offset + k] = ComputeMatchingCostforOnePixel_x_ncc(j, i, d, w, h, hw, hh, imgb, imgm); // DSI[k*w*h + i*w + j]
			}
		}
		else if (2 == costType) // opencv ncc
		{
			for (int d = dmin; d <= dmax; ++d)
			{
				int k = d - dmin; // vertical index

				DSI[offset + k] = ComputeMatchingCostforOnePixel_x_opencvncc(j, i, d, w, h, hw, hh, imgb, imgm); // DSI[k*w*h + i*w + j]
			}
		}
		else
		{

		}
	}
	else if (1==dir) // r2l
	{
		if (0 == costType) // BT
		{
			for (int d = dmin; d <= dmax; ++d)
			{
				int k = d - dmin; // vertical index

				DSI[offset + k] = ComputeMatchingCostforOnePixel_x_BT(j, i, -d, w, h, imgm, imgb); // DSI[k*w*h + i*w + j]
			}
		}
		else if (1 == costType) // ncc
		{
			for (int d = dmin; d <= dmax; ++d)
			{
				int k = d - dmin; // vertical index

				DSI[offset + k] = ComputeMatchingCostforOnePixel_x_ncc(j, i, -d, w, h, hw, hh, imgm, imgb); // DSI[k*w*h + i*w + j]
			}
		}
		else if (2 == costType) // opencv ncc
		{
			for (int d = dmin; d <= dmax; ++d)
			{
				int k = d - dmin; // vertical index

				DSI[offset + k] = ComputeMatchingCostforOnePixel_x_opencvncc(j, i, -d, w, h, hw, hh, imgm, imgb); // DSI[k*w*h + i*w + j]
			}
		}
		else
		{

		}
	}
	else if (2==dir) // u2b
	{
		if (0 == costType) // BT
		{
			for (int d = dmin; d <= dmax; ++d)
			{
				int k = d - dmin; // vertical index

				DSI[offset + k] = ComputeMatchingCostforOnePixel_y_BT(j, i, d, w, h, imgb, imgm); // DSI[k*w*h + i*w + j]
			}
		}
		else if (1 == costType) // ncc
		{
			for (int d = dmin; d <= dmax; ++d)
			{
				int k = d - dmin; // vertical index

				DSI[offset + k] = ComputeMatchingCostforOnePixel_y_ncc(j, i, d, w, h, hw, hh, imgb, imgm); // DSI[k*w*h + i*w + j]
			}
		}
		else if (2 == costType) // opencv ncc
		{
			for (int d = dmin; d <= dmax; ++d)
			{
				int k = d - dmin; // vertical index

				DSI[offset + k] = ComputeMatchingCostforOnePixel_y_opencvncc(j, i, d, w, h, hw, hh, imgb, imgm); // DSI[k*w*h + i*w + j]
			}
		}
		else
		{

		}
	}
	else // b2u
	{
		if (0 == costType) // BT
		{
			for (int d = dmin; d <= dmax; ++d)
			{
				int k = d - dmin; // vertical index

				DSI[offset + k] = ComputeMatchingCostforOnePixel_y_BT(j, i, -d, w, h, imgm, imgb); // DSI[k*w*h + i*w + j]
			}
		}
		else if (1 == costType) // ncc
		{
			for (int d = dmin; d <= dmax; ++d)
			{
				int k = d - dmin; // vertical index

				DSI[offset + k] = ComputeMatchingCostforOnePixel_y_ncc(j, i, -d, w, h, hw, hh, imgm, imgb); // DSI[k*w*h + i*w + j]
			}
		}
		else if (2 == costType) // opencv ncc
		{
			for (int d = dmin; d <= dmax; ++d)
			{
				int k = d - dmin; // vertical index

				DSI[offset + k] = ComputeMatchingCostforOnePixel_y_opencvncc(j, i, -d, w, h, hw, hh, imgm, imgb); // DSI[k*w*h + i*w + j]
			}
		}
		else
		{

		}
	}
}

// 20170102, input stereo images are no need to be transposed
// 20161226, DSI with new layout
// 20161115, four directions: 0:left to right. 1:right to left. 2:up to bottom. 3:bottom to up
// Generate the Disparity Space Image (DSI) based on the pixelwise matching
// cost proposed in <Depth Discontinuities by Pixel-to-Pixel Stereo> by S. Birchfield and C. Tomasi
__global__ void 
kernel_GenerateDSI_l2r_BT(int w, int h,	int nd,			// input: the width and height of stereo images
						  const unsigned char * imgb,	// input: h*w, the rectified grayscale base image
						  const unsigned char * imgm,	// input: h*w, the rectified grayscale matching image
						  int dmin,						// input: scalar, the minimal disparity, can be negative, but make sure dmin<dmax
						  double * DSI					// output:h*w*nd, nd = (dmax - dmin + 1), the output Disparity Space Image
						  )
{
	// the indices
	int k = blockDim.x * blockIdx.x + threadIdx.x; // index of disparity level
	int j = blockDim.y * blockIdx.y + threadIdx.y; 
	int i = blockDim.z * blockIdx.z + threadIdx.z;

	if (k < nd && j < w && i < h)
	{
		int d = dmin + k;

		DSI[(i*w + j)*nd + k] = ComputeMatchingCostforOnePixel_x_BT(j, i, d, w, h, imgb, imgm);
	}
}

// 20170102, input stereo images are no need to be transposed
// 20161226, DSI with new layout
// 20161115, four directions: 0:left to right. 1:right to left. 2:up to bottom. 3:bottom to up
// Generate the Disparity Space Image (DSI) based on the pixelwise matching
// cost proposed in <Depth Discontinuities by Pixel-to-Pixel Stereo> by S. Birchfield and C. Tomasi
__global__ void 
kernel_GenerateDSI_l2r_ncc(int w, int h, int nd,		// input: the width and height of stereo images
						   int hw, int hh,				// the half width and half height of the support window
						   const unsigned char * imgb,	// input: h*w, the rectified grayscale base image
						   const unsigned char * imgm,	// input: h*w, the rectified grayscale matching image
						   int dmin,					// input: scalar, the minimal disparity, can be negative, but make sure dmin<dmax
						   double * DSI					// output:h*w*nd, nd = (dmax - dmin + 1), the output Disparity Space Image
						   )
{
	// the indices
	int k = blockDim.x * blockIdx.x + threadIdx.x; // index of disparity level
	int j = blockDim.y * blockIdx.y + threadIdx.y;
	int i = blockDim.z * blockIdx.z + threadIdx.z;

	if (k < nd && j < w && i < h)
	{
		int d = dmin + k;

		DSI[(i*w + j)*nd + k] = ComputeMatchingCostforOnePixel_x_ncc(j, i, d, w, h, hw, hh, imgb, imgm);
	}
}

// 20170102, input stereo images are no need to be transposed
// 20161226, DSI with new layout
// 20161115, four directions: 0:left to right. 1:right to left. 2:up to bottom. 3:bottom to up
// Generate the Disparity Space Image (DSI) based on the pixelwise matching
// cost proposed in <Depth Discontinuities by Pixel-to-Pixel Stereo> by S. Birchfield and C. Tomasi
__global__ void 
kernel_GenerateDSI_l2r_opencvncc(int w, int h, int nd,		// input: the width and height of stereo images
								 int hw, int hh,			// the half width and half height of the support window
								 const unsigned char * imgb,// input: h*w, the rectified grayscale base image
								 const unsigned char * imgm,// input: h*w, the rectified grayscale matching image
								 int dmin,					// input: scalar, the minimal disparity, can be negative, but make sure dmin<dmax
								 double * DSI				// output:h*w*nd, nd = (dmax - dmin + 1), the output Disparity Space Image
								 )
{
	// the indices
	int k = blockDim.x * blockIdx.x + threadIdx.x; // index of disparity level
	int j = blockDim.y * blockIdx.y + threadIdx.y;
	int i = blockDim.z * blockIdx.z + threadIdx.z;

	if (k < nd && j < w && i < h)
	{
		int d = dmin + k;

		DSI[(i*w + j)*nd + k] = ComputeMatchingCostforOnePixel_x_opencvncc(j, i, d, w, h, hw, hh, imgb, imgm);
	}
}

// 20170102, input stereo images are no need to be transposed
// 20161226, DSI with new layout
// 20161115, four directions: 0:left to right. 1:right to left. 2:up to bottom. 3:bottom to up
// Generate the Disparity Space Image (DSI) based on the pixelwise matching
// cost proposed in <Depth Discontinuities by Pixel-to-Pixel Stereo> by S. Birchfield and C. Tomasi
__global__ void 
kernel_GenerateDSI_r2l_BT(int w, int h,	int nd,			// input: the width and height of stereo images
						  const unsigned char * imgb,	// input: h*w, the rectified grayscale base image
						  const unsigned char * imgm,	// input: h*w, the rectified grayscale matching image
						  int dmin,						// input: scalar, the minimal disparity, can be negative, but make sure dmin<dmax
						  double * DSI					// output:h*w*nd, nd = (dmax - dmin + 1), the output Disparity Space Image
						  )
{
	// the indices
	int k = blockDim.x * blockIdx.x + threadIdx.x; // index of disparity level
	int j = blockDim.y * blockIdx.y + threadIdx.y; 
	int i = blockDim.z * blockIdx.z + threadIdx.z;

	if (k < nd && j < w && i < h)
	{
		int d = dmin + k;

		DSI[(i*w + j)*nd + k] = ComputeMatchingCostforOnePixel_x_BT(j, i, -d, w, h, imgm, imgb);
	}
}

// 20170102, input stereo images are no need to be transposed
// 20161226, DSI with new layout
// 20161115, four directions: 0:left to right. 1:right to left. 2:up to bottom. 3:bottom to up
// Generate the Disparity Space Image (DSI) based on the pixelwise matching
// cost proposed in <Depth Discontinuities by Pixel-to-Pixel Stereo> by S. Birchfield and C. Tomasi
__global__ void 
kernel_GenerateDSI_r2l_ncc(int w, int h, int nd,		// input: the width and height of stereo images
						   int hw, int hh,				// the half width and half height of the support window
						   const unsigned char * imgb,	// input: h*w, the rectified grayscale base image
						   const unsigned char * imgm,	// input: h*w, the rectified grayscale matching image
						   int dmin,					// input: scalar, the minimal disparity, can be negative, but make sure dmin<dmax
						   double * DSI					// output:h*w*nd, nd = (dmax - dmin + 1), the output Disparity Space Image
						   )
{
	// the indices
	int k = blockDim.x * blockIdx.x + threadIdx.x; // index of disparity level
	int j = blockDim.y * blockIdx.y + threadIdx.y; 
	int i = blockDim.z * blockIdx.z + threadIdx.z;

	if (k < nd && j < w && i < h)
	{
		int d = dmin + k;

		DSI[(i*w + j)*nd + k] = ComputeMatchingCostforOnePixel_x_ncc(j, i, -d, w, h, hw, hh, imgm, imgb);
	}
}

// 20170102, input stereo images are no need to be transposed
// 20161226, DSI with new layout
// 20161115, four directions: 0:left to right. 1:right to left. 2:up to bottom. 3:bottom to up
// Generate the Disparity Space Image (DSI) based on the pixelwise matching
// cost proposed in <Depth Discontinuities by Pixel-to-Pixel Stereo> by S. Birchfield and C. Tomasi
__global__ void 
kernel_GenerateDSI_r2l_opencvncc(int w, int h, int nd,		// input: the width and height of stereo images
								 int hw, int hh,			// the half width and half height of the support window
								 const unsigned char * imgb,// input: h*w, the rectified grayscale base image
								 const unsigned char * imgm,// input: h*w, the rectified grayscale matching image
								 int dmin,					// input: scalar, the minimal disparity, can be negative, but make sure dmin<dmax
								 double * DSI				// output:h*w*nd, nd = (dmax - dmin + 1), the output Disparity Space Image
								 )
{
	// the indices
	int k = blockDim.x * blockIdx.x + threadIdx.x; // index of disparity level
	int j = blockDim.y * blockIdx.y + threadIdx.y; 
	int i = blockDim.z * blockIdx.z + threadIdx.z;

	if (k < nd && j < w && i < h)
	{
		int d = dmin + k;

		DSI[(i*w + j)*nd + k] = ComputeMatchingCostforOnePixel_x_opencvncc(j, i, -d, w, h, hw, hh, imgm, imgb);
	}
}

// 20170102, input stereo images are transposed
// 20161226, DSI with new layout
// 20161115, four directions: 0:left to right. 1:right to left. 2:up to bottom. 3:bottom to up
// Generate the Disparity Space Image (DSI) based on the pixelwise matching
// cost proposed in <Depth Discontinuities by Pixel-to-Pixel Stereo> by S. Birchfield and C. Tomasi
__global__ void 
kernel_GenerateDSI_u2b_BT_transposed(int w, int h, int nd,			// input: the width and height of stereo images
									 const unsigned char * imgb,	// input: h*w, the rectified grayscale base image
									 const unsigned char * imgm,	// input: h*w, the rectified grayscale matching image
									 int dmin,						// input: scalar, the minimal disparity, can be negative, but make sure dmin<dmax
									 double * DSI					// output:h*w*nd, nd = (dmax - dmin + 1), the output Disparity Space Image
									 )
{
	// the indices
	int k = blockDim.x * blockIdx.x + threadIdx.x; // index of disparity level
	int j = blockDim.y * blockIdx.y + threadIdx.y; 
	int i = blockDim.z * blockIdx.z + threadIdx.z;

	if (k < nd && j < w && i < h)
	{
		int d = dmin + k;

		DSI[(j*h + i)*nd + k] = ComputeMatchingCostforOnePixel_x_BT(j, i, d, w, h, imgb, imgm);
	}
}

// 20170102, input stereo images are transposed
// 20161226, DSI with new layout
// 20161115, four directions: 0:left to right. 1:right to left. 2:up to bottom. 3:bottom to up
// Generate the Disparity Space Image (DSI) based on the pixelwise matching
// cost proposed in <Depth Discontinuities by Pixel-to-Pixel Stereo> by S. Birchfield and C. Tomasi
__global__ void 
kernel_GenerateDSI_u2b_ncc_transposed(int w, int h, int nd,			// input: the width and height of stereo images
									  int hw, int hh,				// the half width and half height of the support window
									  const unsigned char * imgb,	// input: h*w, the rectified grayscale base image
									  const unsigned char * imgm,	// input: h*w, the rectified grayscale matching image
									  int dmin,						// input: scalar, the minimal disparity, can be negative, but make sure dmin<dmax
									  double * DSI					// output:h*w*nd, nd = (dmax - dmin + 1), the output Disparity Space Image
									  )
{
	// the indices
	int k = blockDim.x * blockIdx.x + threadIdx.x; // index of disparity level
	int j = blockDim.y * blockIdx.y + threadIdx.y; 
	int i = blockDim.z * blockIdx.z + threadIdx.z;

	if (k < nd && j < w && i < h)
	{
		int d = dmin + k;

		DSI[(j*h + i)*nd + k] = ComputeMatchingCostforOnePixel_x_ncc(j, i, d, w, h, hw, hh, imgb, imgm);
	}
}

// 20170102, input stereo images are transposed
// 20161226, DSI with new layout
// 20161115, four directions: 0:left to right. 1:right to left. 2:up to bottom. 3:bottom to up
// Generate the Disparity Space Image (DSI) based on the pixelwise matching
// cost proposed in <Depth Discontinuities by Pixel-to-Pixel Stereo> by S. Birchfield and C. Tomasi
__global__ void 
kernel_GenerateDSI_u2b_opencvncc_transposed(int w, int h, int nd,		// input: the width and height of stereo images
											int hw, int hh,				// the half width and half height of the support window
											const unsigned char * imgb,	// input: h*w, the rectified grayscale base image
											const unsigned char * imgm,	// input: h*w, the rectified grayscale matching image
											int dmin,					// input: scalar, the minimal disparity, can be negative, but make sure dmin<dmax
											double * DSI				// output:h*w*nd, nd = (dmax - dmin + 1), the output Disparity Space Image
											)
{
	// the indices
	int k = blockDim.x * blockIdx.x + threadIdx.x; // index of disparity level
	int j = blockDim.y * blockIdx.y + threadIdx.y; 
	int i = blockDim.z * blockIdx.z + threadIdx.z;

	if (k < nd && j < w && i < h)
	{
		int d = dmin + k;

		DSI[(j*h + i)*nd + k] = ComputeMatchingCostforOnePixel_x_opencvncc(j, i, d, w, h, hw, hh, imgb, imgm);
	}
}

// 20170102, input stereo images are transposed
// 20161226, DSI with new layout
// 20161115, four directions: 0:left to right. 1:right to left. 2:up to bottom. 3:bottom to up
// Generate the Disparity Space Image (DSI) based on the pixelwise matching
// cost proposed in <Depth Discontinuities by Pixel-to-Pixel Stereo> by S. Birchfield and C. Tomasi
__global__ void 
kernel_GenerateDSI_b2u_BT_transposed(int w, int h, int nd,		// input: the width and height of stereo images
									 const unsigned char * imgb,// input: h*w, the rectified grayscale base image
									 const unsigned char * imgm,// input: h*w, the rectified grayscale matching image
									 int dmin,					// input: scalar, the minimal disparity, can be negative, but make sure dmin<dmax
									 double * DSI				// output:h*w*nd, nd = (dmax - dmin + 1), the output Disparity Space Image
									 )
{
	// the indices
	int k = blockDim.x * blockIdx.x + threadIdx.x; // index of disparity level
	int j = blockDim.y * blockIdx.y + threadIdx.y; 
	int i = blockDim.z * blockIdx.z + threadIdx.z;

	if (k < nd && j < w && i < h)
	{
		int d = dmin + k;

		DSI[(j*h + i)*nd + k] = ComputeMatchingCostforOnePixel_x_BT(j, i, -d, w, h, imgm, imgb);
	}
}

// 20170102, input stereo images are transposed
// 20161226, DSI with new layout
// 20161115, four directions: 0:left to right. 1:right to left. 2:up to bottom. 3:bottom to up
// Generate the Disparity Space Image (DSI) based on the pixelwise matching
// cost proposed in <Depth Discontinuities by Pixel-to-Pixel Stereo> by S. Birchfield and C. Tomasi
__global__ void 
kernel_GenerateDSI_b2u_ncc_transposed(int w, int h, int nd,			// input: the width and height of stereo images
									  int hw, int hh,				// the half width and half height of the support window
									  const unsigned char * imgb,	// input: h*w, the rectified grayscale base image
									  const unsigned char * imgm,	// input: h*w, the rectified grayscale matching image
									  int dmin,						// input: scalar, the minimal disparity, can be negative, but make sure dmin<dmax
									  double * DSI					// output:h*w*nd, nd = (dmax - dmin + 1), the output Disparity Space Image
									  )
{
	// the indices
	int k = blockDim.x * blockIdx.x + threadIdx.x; // index of disparity level
	int j = blockDim.y * blockIdx.y + threadIdx.y; 
	int i = blockDim.z * blockIdx.z + threadIdx.z;

	if (k < nd && j < w && i < h)
	{
		int d = dmin + k;

		DSI[(j*h + i)*nd + k] = ComputeMatchingCostforOnePixel_x_ncc(j, i, -d, w, h, hw, hh, imgm, imgb);
	}
}

// 20170102, input stereo images are transposed
// 20161226, DSI with new layout
// 20161115, four directions: 0:left to right. 1:right to left. 2:up to bottom. 3:bottom to up
// Generate the Disparity Space Image (DSI) based on the pixelwise matching
// cost proposed in <Depth Discontinuities by Pixel-to-Pixel Stereo> by S. Birchfield and C. Tomasi
__global__ void 
kernel_GenerateDSI_b2u_opencvncc_transposed(int w, int h, int nd,		// input: the width and height of stereo images
											int hw, int hh,				// the half width and half height of the support window
											const unsigned char * imgb,	// input: h*w, the rectified grayscale base image
											const unsigned char * imgm,	// input: h*w, the rectified grayscale matching image
											int dmin,					// input: scalar, the minimal disparity, can be negative, but make sure dmin<dmax
											double * DSI				// output:h*w*nd, nd = (dmax - dmin + 1), the output Disparity Space Image
											)
{
	// the indices
	int k = blockDim.x * blockIdx.x + threadIdx.x; // index of disparity level
	int j = blockDim.y * blockIdx.y + threadIdx.y; 
	int i = blockDim.z * blockIdx.z + threadIdx.z;

	if (k < nd && j < w && i < h)
	{
		int d = dmin + k;

		DSI[(j*h + i)*nd + k] = ComputeMatchingCostforOnePixel_x_opencvncc(j, i, -d, w, h, hw, hh, imgm, imgb);
	}
}

// 20170102, input stereo images are transposed
// 20161226, DSI with new layout
// 20161115, four directions: 0:left to right. 1:right to left. 2:up to bottom. 3:bottom to up
// Generate the Disparity Space Image (DSI) based on the pixelwise matching
// cost proposed in <Depth Discontinuities by Pixel-to-Pixel Stereo> by S. Birchfield and C. Tomasi
//__global__ void 
//kernel_GenerateDSI_l2r_BT_transposed(int w, int h,	int nd,			// input: the width and height of stereo images
//									 const unsigned char * imgb,	// input: h*w, the rectified grayscale base image
//									 const unsigned char * imgm,	// input: h*w, the rectified grayscale matching image
//									 int dmin,						// input: scalar, the minimal disparity, can be negative, but make sure dmin<dmax
//									 double * DSI					// output:h*w*nd, nd = (dmax - dmin + 1), the output Disparity Space Image
//									 )
//{
//	// the indices
//	int k = blockDim.x * blockIdx.x + threadIdx.x; // index of disparity level
//	int j = blockDim.y * blockIdx.y + threadIdx.y; 
//	int i = blockDim.z * blockIdx.z + threadIdx.z;
//
//	if (k < nd && j < w && i < h)
//	{
//		int d = dmin + k;
//
//		DSI[(j*h + i)*nd + k] = ComputeMatchingCostforOnePixel_x_BT(j, i, d, w, h, imgb, imgm);
//	}
//}

// 20161115, four directions: 0:left to right. 1:right to left. 2:up to bottom. 3:bottom to up
// Generate the Disparity Space Image (DSI) based on the pixelwise matching
// cost proposed in <Depth Discontinuities by Pixel-to-Pixel Stereo> by S. Birchfield and C. Tomasi
__global__ void 
kernel_CostAggregation_OneParallelLine(int w, int h, int nd,	// input: the width and height of stereo images
									   double * C,				// output:h*w*nd, nd = (dmax - dmin + 1), the output Disparity Space Image
									   double P1,				// input: constant penalty pixels in the neigborhood of (x,y), for which the disparity changes a little bit (that is 1 pixel)
									   double P2,				// input: a larger constant penalty for all larger disparity changes
									   int idx1,				// input: the row index of the parallel row when bParallelRow is true, otherwise, is the colomn index of the parallel colomn
									   bool bParallelRow,		// input: true: pixels in a row are processed in parallel, false: pixels in a colomn are processed in parallel
									   int dir					// input: 0:(0,-1), 1:(1,-1), 2:(1,0), 3:(1,1), 4:(0,1), 5:(-1,1), 6:(-1,0), 7:(-1,-1)
									   )
{
	extern __shared__ double s_pre[]; // dynamic shared memory of aggregated costs of all previous pixels within the same thread block

	if (idx1 < 0) // no matter what, idx1 has to be non-negative
	{
		return;
	}

	// the colomn index of current pixel in case bParallelRow is true, otherwise, it's the row index of current pixel in case bParallelRow is false
	int idx2 = blockDim.x * blockIdx.x + threadIdx.x;

	if (bParallelRow) // it's a parallel row
	{
		if (2 == dir || 6 == dir) // horizontal propagation is not applicable in parallel row form
		{
			return;
		}

		if (idx1 >= h || idx2 >= w) // cross the boundary
		{
			return;
		}

		// image indices of current pixel
		int i = idx1;
		int j = idx2;

		// image indices of previous pixel according to the propagation direction
		int i_pre, j_pre;
		previous_ij(i, j, i_pre, j_pre, dir);

		if (i_pre < 0 || i_pre >= h || j_pre < 0 || j_pre >= w) // previous pixel cross the boundary
		{
			return;
		}

//		CostAggregation_OnePixel(w, h, nd, C, i, j, i_pre, j_pre, P1, P2);
		
		///////////////////////////////////////////////////////////////
		/*const int nnn = 100;

		double cur[nnn], pre[nnn];

		int nPix = w*h;
		int wi_j_cur = w*i + j;
		int wi_j_pre = w*i_pre + j_pre;

		for (int k = 0; k < nd; ++k)
		{
			cur[k] = C[nPix*k + wi_j_cur];
			pre[k] = C[nPix*k + wi_j_pre];
		}

		CostAggregation_OnePixel(cur, pre, nd, P1, P2);

		for (int k = 0; k < nd; ++k)
		{
			C[nPix*k + wi_j_cur] = cur[k];
		}*/

		////////////////////////////////////////////////////////////////
		int nPix = w*h;
		int wi_j_pre = w*i_pre + j_pre;
		int inxd = threadIdx.x*nd;

		for (int k = 0; k < nd; ++k)
		{
			s_pre[inxd + k] = C[nPix*k + wi_j_pre];
		}

		CostAggregation_OnePixel(w, h, nd, threadIdx.x, s_pre, C, i, j, P1, P2);
	}
	else // it's a parallel colomn
	{
		if (0 == dir || 4 == dir) // vertical propagation is not applicable in parallel colomn form
		{
			return;
		}

		if (idx1 >= w || idx2 >= h) // cross the boundary
		{
			return;
		}

		// image indices of current pixel
		int i = idx2;
		int j = idx1;

		// image indices of previous pixel according to the propagation direction
		int i_pre, j_pre;
		previous_ij(i, j, i_pre, j_pre, dir);

		if (i_pre < 0 || i_pre >= h || j_pre < 0 || j_pre >= w) // previous pixel cross the boundary
		{
			return;
		}

//		CostAggregation_OnePixel(w, h, nd, C, i, j, i_pre, j_pre, P1, P2);

		////////////////////////////////////////////////////////////////////////
		/*const int nnn = 100;

		double cur[nnn], pre[nnn];

		int nPix = w*h;
		int wi_j_cur = w*i + j;
		int wi_j_pre = w*i_pre + j_pre;

		for (int k = 0; k < nd; ++k)
		{
			cur[k] = C[nPix*k + wi_j_cur];
			pre[k] = C[nPix*k + wi_j_pre];
		}

		CostAggregation_OnePixel(cur, pre, nd, P1, P2);

		for (int k = 0; k < nd; ++k)
		{
			C[nPix*k + wi_j_cur] = cur[k];
		}*/

		///////////////////////////////////////////////////////////////////////////
		int nPix = w*h;
		int wi_j_pre = w*i_pre + j_pre;
		int inxd = threadIdx.x*nd;

		for (int k = 0; k < nd; ++k)
		{
			s_pre[inxd + k] = C[nPix*k + wi_j_pre];
		}

		CostAggregation_OnePixel(w, h, nd, threadIdx.x, s_pre, C, i, j, P1, P2);
	}
}

// 20161226, propagate from up pixel to current pixel
// 20161115, four directions: 0:left to right. 1:right to left. 2:up to bottom. 3:bottom to up
// Generate the Disparity Space Image (DSI) based on the pixelwise matching
// cost proposed in <Depth Discontinuities by Pixel-to-Pixel Stereo> by S. Birchfield and C. Tomasi
__global__ void 
kernel_CostAggregation_OneParallelRow_Up(int w, int h, int nd,	// input: the width and height of stereo images
										 double * C,			// output:h*w*nd, nd = (dmax - dmin + 1), the output Disparity Space Image
										 double P1,				// input: constant penalty pixels in the neigborhood of (x,y), for which the disparity changes a little bit (that is 1 pixel)
										 double P2,				// input: a larger constant penalty for all larger disparity changes
										 int i					// input: the row index of current parallel row, should be 1<=i<h
										 )
{
	// the colomn index of current pixel
//	int j = blockDim.x * blockIdx.x + threadIdx.x;

	int k = blockDim.x * blockIdx.x + threadIdx.x;
	int j = blockDim.y * blockIdx.y + threadIdx.y;

	if (j < w && k < nd) // j should be 0<=j<w
	{
//		CostAggregation_OnePixel_new(w, h, nd, C, i, j, i - 1, j, P1, P2);
		CostAggregation_OnePixel_new(w, h, nd, C, i, j, k, i - 1, j, P1, P2);
	}
}

// 20161226, propagate from up right pixel to current pixel
// 20161115, four directions: 0:left to right. 1:right to left. 2:up to bottom. 3:bottom to up
// Generate the Disparity Space Image (DSI) based on the pixelwise matching
// cost proposed in <Depth Discontinuities by Pixel-to-Pixel Stereo> by S. Birchfield and C. Tomasi
__global__ void 
kernel_CostAggregation_OneParallelRow_UpRight(int w, int h, int nd,	// input: the width and height of stereo images
											  double * C,			// output:h*w*nd, nd = (dmax - dmin + 1), the output Disparity Space Image
											  double P1,			// input: constant penalty pixels in the neigborhood of (x,y), for which the disparity changes a little bit (that is 1 pixel)
											  double P2,			// input: a larger constant penalty for all larger disparity changes
											  int i					// input: the row index of current parallel row, should be 1<=i<h
											  )
{
	// the colomn index of current pixel
//	int j = blockDim.x * blockIdx.x + threadIdx.x;

	int k = blockDim.x * blockIdx.x + threadIdx.x;
	int j = blockDim.y * blockIdx.y + threadIdx.y;

	if (j < (w - 1) && k < nd) // j should be 0<=j<(w-1)
	{
//		CostAggregation_OnePixel_new(w, h, nd, C, i, j, i - 1, j + 1, P1, P2);
		CostAggregation_OnePixel_new(w, h, nd, C, i, j, k, i - 1, j + 1, P1, P2);
	}
}

// 20161226, propagate from up left pixel to current pixel
// 20161115, four directions: 0:left to right. 1:right to left. 2:up to bottom. 3:bottom to up
// Generate the Disparity Space Image (DSI) based on the pixelwise matching
// cost proposed in <Depth Discontinuities by Pixel-to-Pixel Stereo> by S. Birchfield and C. Tomasi
__global__ void 
kernel_CostAggregation_OneParallelRow_UpLeft(int w, int h, int nd,	// input: the width and height of stereo images
											 double * C,			// output:h*w*nd, nd = (dmax - dmin + 1), the output Disparity Space Image
											 double P1,				// input: constant penalty pixels in the neigborhood of (x,y), for which the disparity changes a little bit (that is 1 pixel)
											 double P2,				// input: a larger constant penalty for all larger disparity changes
											 int i					// input: the row index of current parallel row, should be 1<=i<h
											 )
{
	// the colomn index of current pixel
//	int j = blockDim.x * blockIdx.x + threadIdx.x;

	int k = blockDim.x * blockIdx.x + threadIdx.x;
	int j = blockDim.y * blockIdx.y + threadIdx.y;

	if (j > 0 && j < w && k < nd) // j should be 0<j<w
	{
//		CostAggregation_OnePixel_new(w, h, nd, C, i, j, i - 1, j - 1, P1, P2);
		CostAggregation_OnePixel_new(w, h, nd, C, i, j, k, i - 1, j - 1, P1, P2);
	}
}

// 20161226, propagate from bottom pixel to current pixel
// 20161115, four directions: 0:left to right. 1:right to left. 2:up to bottom. 3:bottom to up
// Generate the Disparity Space Image (DSI) based on the pixelwise matching
// cost proposed in <Depth Discontinuities by Pixel-to-Pixel Stereo> by S. Birchfield and C. Tomasi
__global__ void 
kernel_CostAggregation_OneParallelRow_Bottom(int w, int h, int nd,	// input: the width and height of stereo images
											 double * C,			// output:h*w*nd, nd = (dmax - dmin + 1), the output Disparity Space Image
											 double P1,				// input: constant penalty pixels in the neigborhood of (x,y), for which the disparity changes a little bit (that is 1 pixel)
											 double P2,				// input: a larger constant penalty for all larger disparity changes
											 int i					// input: the row index of current parallel row, should be 0<=i<(h-1)
											 )
{
	// the colomn index of current pixel
//	int j = blockDim.x * blockIdx.x + threadIdx.x;

	int k = blockDim.x * blockIdx.x + threadIdx.x;
	int j = blockDim.y * blockIdx.y + threadIdx.y;

	if (j < w && k < nd) // j should be 0<=j<w
	{
//		CostAggregation_OnePixel_new(w, h, nd, C, i, j, i + 1, j, P1, P2);
		CostAggregation_OnePixel_new(w, h, nd, C, i, j, k, i + 1, j, P1, P2);
	}
}

// 20161226, propagate from bottom right pixel to current pixel
// 20161115, four directions: 0:left to right. 1:right to left. 2:up to bottom. 3:bottom to up
// Generate the Disparity Space Image (DSI) based on the pixelwise matching
// cost proposed in <Depth Discontinuities by Pixel-to-Pixel Stereo> by S. Birchfield and C. Tomasi
__global__ void 
kernel_CostAggregation_OneParallelRow_BottomRight(int w, int h, int nd,	// input: the width and height of stereo images
												  double * C,			// output:h*w*nd, nd = (dmax - dmin + 1), the output Disparity Space Image
												  double P1,			// input: constant penalty pixels in the neigborhood of (x,y), for which the disparity changes a little bit (that is 1 pixel)
												  double P2,			// input: a larger constant penalty for all larger disparity changes
												  int i					// input: the row index of current parallel row, should be 0<=i<(h-1)
												  )
{
	// the colomn index of current pixel
//	int j = blockDim.x * blockIdx.x + threadIdx.x;

	int k = blockDim.x * blockIdx.x + threadIdx.x;
	int j = blockDim.y * blockIdx.y + threadIdx.y;

	if (j < (w - 1) && k < nd) // j should be 0<=j<(w-1)
	{
//		CostAggregation_OnePixel_new(w, h, nd, C, i, j, i + 1, j + 1, P1, P2);
		CostAggregation_OnePixel_new(w, h, nd, C, i, j, k, i + 1, j + 1, P1, P2);
	}
}

// 20161226, propagate from bottom left pixel to current pixel
// 20161115, four directions: 0:left to right. 1:right to left. 2:up to bottom. 3:bottom to up
// Generate the Disparity Space Image (DSI) based on the pixelwise matching
// cost proposed in <Depth Discontinuities by Pixel-to-Pixel Stereo> by S. Birchfield and C. Tomasi
__global__ void 
kernel_CostAggregation_OneParallelRow_BottomLeft(int w, int h, int nd,	// input: the width and height of stereo images
												 double * C,			// output:h*w*nd, nd = (dmax - dmin + 1), the output Disparity Space Image
												 double P1,				// input: constant penalty pixels in the neigborhood of (x,y), for which the disparity changes a little bit (that is 1 pixel)
												 double P2,				// input: a larger constant penalty for all larger disparity changes
												 int i					// input: the row index of current parallel row, should be 0<=i<(h-1)
												 )
{
	// the colomn index of current pixel
//	int j = blockDim.x * blockIdx.x + threadIdx.x;

	int k = blockDim.x * blockIdx.x + threadIdx.x;
	int j = blockDim.y * blockIdx.y + threadIdx.y;

	if (j > 0 && j < w && k < nd) // j should be 0<j<w
	{
//		CostAggregation_OnePixel_new(w, h, nd, C, i, j, i + 1, j - 1, P1, P2);
		CostAggregation_OnePixel_new(w, h, nd, C, i, j, k, i + 1, j - 1, P1, P2);
	}
}

// 20161226, propagate from right pixel to current pixel
// 20161115, four directions: 0:left to right. 1:right to left. 2:up to bottom. 3:bottom to up
// Generate the Disparity Space Image (DSI) based on the pixelwise matching
// cost proposed in <Depth Discontinuities by Pixel-to-Pixel Stereo> by S. Birchfield and C. Tomasi
__global__ void 
kernel_CostAggregation_OneParallelCol_Right(int w, int h, int nd,	// input: the width and height of stereo images
											double * C,				// output:h*w*nd, nd = (dmax - dmin + 1), the output Disparity Space Image
											double P1,				// input: constant penalty pixels in the neigborhood of (x,y), for which the disparity changes a little bit (that is 1 pixel)
											double P2,				// input: a larger constant penalty for all larger disparity changes
											int j					// input: the column index of current parallel column, should be 0<=j<(w-1)
											)
{
	// the colomn index of current pixel
//	int i = blockDim.x * blockIdx.x + threadIdx.x;

	int k = blockDim.x * blockIdx.x + threadIdx.x;
	int i = blockDim.y * blockIdx.y + threadIdx.y;

	if (i < h && k < nd) // i should be 0<=i<h
	{
//		CostAggregation_OnePixel_new(w, h, nd, C, i, j, i, j + 1, P1, P2);
		CostAggregation_OnePixel_new(w, h, nd, C, i, j, k, i, j + 1, P1, P2);
	}
}

// 20161226, propagate from up right pixel to current pixel
// 20161115, four directions: 0:left to right. 1:right to left. 2:up to bottom. 3:bottom to up
// Generate the Disparity Space Image (DSI) based on the pixelwise matching
// cost proposed in <Depth Discontinuities by Pixel-to-Pixel Stereo> by S. Birchfield and C. Tomasi
__global__ void 
kernel_CostAggregation_OneParallelCol_UpRight(int w, int h, int nd,	// input: the width and height of stereo images
											  double * C,			// output:h*w*nd, nd = (dmax - dmin + 1), the output Disparity Space Image
											  double P1,			// input: constant penalty pixels in the neigborhood of (x,y), for which the disparity changes a little bit (that is 1 pixel)
											  double P2,			// input: a larger constant penalty for all larger disparity changes
											  int j					// input: the column index of current parallel column, should be 0<=j<(w-1)
											  )
{
	// the colomn index of current pixel
//	int i = blockDim.x * blockIdx.x + threadIdx.x;

	int k = blockDim.x * blockIdx.x + threadIdx.x;
	int i = blockDim.y * blockIdx.y + threadIdx.y;

	if (i > 0 && i < h && k < nd) // i should be 0<i<h
	{
//		CostAggregation_OnePixel_new(w, h, nd, C, i, j, i - 1, j + 1, P1, P2);
		CostAggregation_OnePixel_new(w, h, nd, C, i, j, k, i - 1, j + 1, P1, P2);
	}
}

// 20161226, propagate from bottom right pixel to current pixel
// 20161115, four directions: 0:left to right. 1:right to left. 2:up to bottom. 3:bottom to up
// Generate the Disparity Space Image (DSI) based on the pixelwise matching
// cost proposed in <Depth Discontinuities by Pixel-to-Pixel Stereo> by S. Birchfield and C. Tomasi
__global__ void 
kernel_CostAggregation_OneParallelCol_BottomRight(int w, int h, int nd,	// input: the width and height of stereo images
												  double * C,			// output:h*w*nd, nd = (dmax - dmin + 1), the output Disparity Space Image
												  double P1,			// input: constant penalty pixels in the neigborhood of (x,y), for which the disparity changes a little bit (that is 1 pixel)
												  double P2,			// input: a larger constant penalty for all larger disparity changes
												  int j					// input: the column index of current parallel column, should be 0<=j<(w-1)
												  )
{
	// the colomn index of current pixel
//	int i = blockDim.x * blockIdx.x + threadIdx.x;

	int k = blockDim.x * blockIdx.x + threadIdx.x;
	int i = blockDim.y * blockIdx.y + threadIdx.y;

	if (i < (h - 1) && k < nd) // i should be 0<=i<(h-1)
	{
//		CostAggregation_OnePixel_new(w, h, nd, C, i, j, i + 1, j + 1, P1, P2);
		CostAggregation_OnePixel_new(w, h, nd, C, i, j, k, i + 1, j + 1, P1, P2);
	}
}

// 20161226, propagate from left pixel to current pixel
// 20161115, four directions: 0:left to right. 1:right to left. 2:up to bottom. 3:bottom to up
// Generate the Disparity Space Image (DSI) based on the pixelwise matching
// cost proposed in <Depth Discontinuities by Pixel-to-Pixel Stereo> by S. Birchfield and C. Tomasi
__global__ void 
kernel_CostAggregation_OneParallelCol_Left(int w, int h, int nd,// input: the width and height of stereo images
										   double * C,			// output:h*w*nd, nd = (dmax - dmin + 1), the output Disparity Space Image
										   double P1,			// input: constant penalty pixels in the neigborhood of (x,y), for which the disparity changes a little bit (that is 1 pixel)
										   double P2,			// input: a larger constant penalty for all larger disparity changes
										   int j				// input: the column index of current parallel column, should be 0<j<w
										   )
{
	// the colomn index of current pixel
//	int i = blockDim.x * blockIdx.x + threadIdx.x;

	int k = blockDim.x * blockIdx.x + threadIdx.x;
	int i = blockDim.y * blockIdx.y + threadIdx.y;

	if (i < h && k < nd) // i should be 0<=i<h
	{
//		CostAggregation_OnePixel_new(w, h, nd, C, i, j, i, j - 1, P1, P2);
		CostAggregation_OnePixel_new(w, h, nd, C, i, j, k, i, j - 1, P1, P2);
	}
}

// 20161226, propagate from up left pixel to current pixel
// 20161115, four directions: 0:left to right. 1:right to left. 2:up to bottom. 3:bottom to up
// Generate the Disparity Space Image (DSI) based on the pixelwise matching
// cost proposed in <Depth Discontinuities by Pixel-to-Pixel Stereo> by S. Birchfield and C. Tomasi
__global__ void 
kernel_CostAggregation_OneParallelCol_UpLeft(int w, int h, int nd,	// input: the width and height of stereo images
											 double * C,			// output:h*w*nd, nd = (dmax - dmin + 1), the output Disparity Space Image
											 double P1,				// input: constant penalty pixels in the neigborhood of (x,y), for which the disparity changes a little bit (that is 1 pixel)
											 double P2,				// input: a larger constant penalty for all larger disparity changes
											 int j					// input: the column index of current parallel column, should be 0<j<w
											 )
{
	// the colomn index of current pixel
//	int i = blockDim.x * blockIdx.x + threadIdx.x;

	int k = blockDim.x * blockIdx.x + threadIdx.x;
	int i = blockDim.y * blockIdx.y + threadIdx.y;

	if (i > 0 && i < h && k < nd) // i should be 0<=i<h
	{
//		CostAggregation_OnePixel_new(w, h, nd, C, i, j, i - 1, j - 1, P1, P2);
		CostAggregation_OnePixel_new(w, h, nd, C, i, j, k, i - 1, j - 1, P1, P2);
	}
}

// 20161226, propagate from bottom left pixel to current pixel
// 20161115, four directions: 0:left to right. 1:right to left. 2:up to bottom. 3:bottom to up
// Generate the Disparity Space Image (DSI) based on the pixelwise matching
// cost proposed in <Depth Discontinuities by Pixel-to-Pixel Stereo> by S. Birchfield and C. Tomasi
__global__ void 
kernel_CostAggregation_OneParallelCol_BottomLeft(int w, int h, int nd,	// input: the width and height of stereo images
												 double * C,			// output:h*w*nd, nd = (dmax - dmin + 1), the output Disparity Space Image
												 double P1,				// input: constant penalty pixels in the neigborhood of (x,y), for which the disparity changes a little bit (that is 1 pixel)
												 double P2,				// input: a larger constant penalty for all larger disparity changes
												 int j					// input: the column index of current parallel column, should be 0<j<w
												 )
{
	// the colomn index of current pixel
//	int i = blockDim.x * blockIdx.x + threadIdx.x;

	int k = blockDim.x * blockIdx.x + threadIdx.x;
	int i = blockDim.y * blockIdx.y + threadIdx.y;

	if (i < (h - 1) && k < nd) // i should be 0<=i<h
	{
//		CostAggregation_OnePixel_new(w, h, nd, C, i, j, i + 1, j - 1, P1, P2);
		CostAggregation_OnePixel_new(w, h, nd, C, i, j, k, i + 1, j - 1, P1, P2);
	}
}

// 20161225, Merry Christmas to the world!
// the parallel row moves from up to bottom
// the previous pixel is right above current pixel
//__global__ void 
//kernel_CostAggregation_ParallelRow_up2bottom_0(int w, int h, int nd,// input: the width and height of stereo images
//											   double * C,			// output:h*w*nd, nd = (dmax - dmin + 1), the output Disparity Space Image
//											   double P1,			// input: constant penalty pixels in the neigborhood of (x,y), for which the disparity changes a little bit (that is 1 pixel)
//											   double P2,			// input: a larger constant penalty for all larger disparity changes
//											   int dir				// input: 0:(0,-1), 1:(1,-1), 2:(1,0), 3:(1,1), 4:(0,1), 5:(-1,1), 6:(-1,0), 7:(-1,-1)
//											   )
//{
//	extern __shared__ double s_pre[]; // dynamic shared memory of aggregated costs of all previous pixels within the same thread block
//
//	// the colomn index of current pixel
//	int j = blockDim.x * blockIdx.x + threadIdx.x;
//
//	if (j < w)
//	{
//		for (int i = 1; i < h; ++i)
//		{
//			// indices of previous pixel on the propagation direction
//			int i_pre = i - 1;
//			int j_pre = j;
//			
//			int nPix = w*h;
//			int wi_j_pre = w*i_pre + j_pre;
//			int inxd = threadIdx.x*nd;
//
//			for (int k = 0; k < nd; ++k)
//			{
//				s_pre[inxd + k] = C[nPix*k + wi_j_pre];
//			}
//
//			CostAggregation_OnePixel(w, h, nd, threadIdx.x, s_pre, C, i, j, P1, P2);
//		}
//
//		// image indices of current pixel
//		int i = idx1;
//		int j = idx2;
//
//		// image indices of previous pixel according to the propagation direction
//		int i_pre, j_pre;
//		previous_ij(i, j, i_pre, j_pre, dir);
//
//		if (i_pre < 0 || i_pre >= h || j_pre < 0 || j_pre >= w) // previous pixel cross the boundary
//		{
//			return;
//		}
//
////		CostAggregation_OnePixel(w, h, nd, C, i, j, i_pre, j_pre, P1, P2);
//		
//		///////////////////////////////////////////////////////////////
//		/*const int nnn = 100;
//
//		double cur[nnn], pre[nnn];
//
//		int nPix = w*h;
//		int wi_j_cur = w*i + j;
//		int wi_j_pre = w*i_pre + j_pre;
//
//		for (int k = 0; k < nd; ++k)
//		{
//			cur[k] = C[nPix*k + wi_j_cur];
//			pre[k] = C[nPix*k + wi_j_pre];
//		}
//
//		CostAggregation_OnePixel(cur, pre, nd, P1, P2);
//
//		for (int k = 0; k < nd; ++k)
//		{
//			C[nPix*k + wi_j_cur] = cur[k];
//		}*/
//
//		////////////////////////////////////////////////////////////////
//		int nPix = w*h;
//		int wi_j_pre = w*i_pre + j_pre;
//		int inxd = threadIdx.x*nd;
//
//		for (int k = 0; k < nd; ++k)
//		{
//			s_pre[inxd + k] = C[nPix*k + wi_j_pre];
//		}
//
//		CostAggregation_OnePixel(w, h, nd, threadIdx.x, s_pre, C, i, j, P1, P2);
//	}
//	else // it's a parallel colomn
//	{
//		if (0 == dir || 4 == dir) // vertical propagation is not applicable in parallel colomn form
//		{
//			return;
//		}
//
//		if (idx1 >= w || idx2 >= h) // cross the boundary
//		{
//			return;
//		}
//
//		// image indices of current pixel
//		int i = idx2;
//		int j = idx1;
//
//		// image indices of previous pixel according to the propagation direction
//		int i_pre, j_pre;
//		previous_ij(i, j, i_pre, j_pre, dir);
//
//		if (i_pre < 0 || i_pre >= h || j_pre < 0 || j_pre >= w) // previous pixel cross the boundary
//		{
//			return;
//		}
//
////		CostAggregation_OnePixel(w, h, nd, C, i, j, i_pre, j_pre, P1, P2);
//
//		////////////////////////////////////////////////////////////////////////
//		/*const int nnn = 100;
//
//		double cur[nnn], pre[nnn];
//
//		int nPix = w*h;
//		int wi_j_cur = w*i + j;
//		int wi_j_pre = w*i_pre + j_pre;
//
//		for (int k = 0; k < nd; ++k)
//		{
//			cur[k] = C[nPix*k + wi_j_cur];
//			pre[k] = C[nPix*k + wi_j_pre];
//		}
//
//		CostAggregation_OnePixel(cur, pre, nd, P1, P2);
//
//		for (int k = 0; k < nd; ++k)
//		{
//			C[nPix*k + wi_j_cur] = cur[k];
//		}*/
//
//		///////////////////////////////////////////////////////////////////////////
//		int nPix = w*h;
//		int wi_j_pre = w*i_pre + j_pre;
//		int inxd = threadIdx.x*nd;
//
//		for (int k = 0; k < nd; ++k)
//		{
//			s_pre[inxd + k] = C[nPix*k + wi_j_pre];
//		}
//
//		CostAggregation_OnePixel(w, h, nd, threadIdx.x, s_pre, C, i, j, P1, P2);
//	}
//}

__global__ void 
kernel_AddVec_double(double * A,			// input & output: A[i] += B[i];
					 const double * B,		// input: 
					 int n					// input: number of elements
					 )
{
	// the colomn index of current pixel in case bParallelRow is true, otherwise, it's the row index of current pixel in case bParallelRow is false
	int i = blockDim.x * blockIdx.x + threadIdx.x;

	if (i < n)
	{
		A[i] += B[i];
	}
}

// transpose a 2D array
__global__ void 
kernel_transpose_double(int w, int h,		// input: the width and height of the 2d array
						double * out,		// output:the transposed 2d array
						const double * in	// input: the original 2d array
						)
{
	int i = blockDim.y*blockIdx.y + threadIdx.y;
	int j = blockDim.x*blockIdx.x + threadIdx.x;

	if (i < h && j < w)
	{
		out[j*h + i] = in[i*w + j]; // obviously, reading from in array is coalesced, writing to out array is not.
	}
}

// transpose a 2D array
__global__ void 
kernel_transpose_uchar(int w, int h,			// input: the width and height of the 2d array
					   unsigned char * out,		// output:the transposed 2d array
					   const unsigned char * in	// input: the original 2d array
					   )
{
	int i = blockDim.y*blockIdx.y + threadIdx.y;
	int j = blockDim.x*blockIdx.x + threadIdx.x;

	if (i < h && j < w)
	{
		out[j*h + i] = in[i*w + j]; // obviously, reading from in array is coalesced, writing to out array is not.
	}
}

// transpose a 2D array
// the thread block should be a square otherwise the result will likely be wrong
__global__ void 
kernel_transpose_tile_double(int w, int h,		// input: the width and height of the 2d array
							 double * out,		// output:the transposed 2d array
							 const double * in	// input: the original 2d array
							 )
{
	extern __shared__ double s_tile[];

	int i_in_corner = blockDim.y*blockIdx.y;
	int j_in_corner = blockDim.x*blockIdx.x;

	int i_out_corner = j_in_corner;
	int j_out_corner = i_in_corner;

	int si = threadIdx.y;
	int sj = threadIdx.x;

	int i_in = i_in_corner + si;
	int j_in = j_in_corner + sj;
	
	if (i_in < h && j_in < w)
	{
		s_tile[sj*blockDim.y + si] = in[i_in*w + j_in];
	}
	__syncthreads();

	int i_out = i_out_corner + si;
	int j_out = j_out_corner + sj;

	if (i_out < w && j_out < h)
	{
		out[i_out*h + j_out] = s_tile[si*blockDim.x + sj];
	}
}

// transpose a 2D array
// the thread block should be a square otherwise the result will likely be wrong
__global__ void 
kernel_transpose_tile_uchar(int w, int h,				// input: the width and height of the 2d array
							unsigned char * out,		// output:the transposed 2d array
							const unsigned char * in	// input: the original 2d array
							)
{
	extern __shared__ unsigned char s_square[];

	int i_in_corner = blockDim.y*blockIdx.y;
	int j_in_corner = blockDim.x*blockIdx.x;

	int i_out_corner = j_in_corner;
	int j_out_corner = i_in_corner;

	int si = threadIdx.y;
	int sj = threadIdx.x;

	int i_in = i_in_corner + si;
	int j_in = j_in_corner + sj;
	
	if (i_in < h && j_in < w)
	{
		s_square[sj*blockDim.y + si] = in[i_in*w + j_in];
	}
	__syncthreads();

	int i_out = i_out_corner + si;
	int j_out = j_out_corner + sj;

	if (i_out < w && j_out < h)
	{
		out[i_out*h + j_out] = s_square[si*blockDim.x + sj];
	}
}

// 20170115
// setup the random states for all the elements in the even 2D field
__global__ void 
kernel_setup_randstates_even(hiprandState * states,		// output:the array of all the random states
							 int w, int h, int wo,		// input: the width, height of the even 2D field and the original width of the original image
							 unsigned long long seed	// input: the seed
							 )
{
	int i = blockDim.y*blockIdx.y + threadIdx.y;
	int j = blockDim.x*blockIdx.x + threadIdx.x;

	if (i < h && j < w)
	{
		int id = i*w + j; // the ID of the element in the even field

		// the original indices in the original 2D field
		int i2 = 2 * i;
		int j2 = 2 * j;

		int ido = i2*wo + j2; // the ID of the element in the original field

		// Each thread gets same seed, a different sequence number, no offset
		hiprand_init(seed, ido, 0, &states[id]);
	}
}

// 20170115
// setup the random states for all the elements in the odd 2D field
__global__ void 
kernel_setup_randstates_odd(hiprandState * states,	// output:the array of all the random states
							int w, int h, int wo,	// input: the width, height of the odd 2D field and the original width of the original image
							unsigned long long seed	// input: the seed
							)
{
	int i = blockDim.y*blockIdx.y + threadIdx.y;
	int j = blockDim.x*blockIdx.x + threadIdx.x;

	if (i < h && j < w)
	{
		int id = i*w + j; // the ID of the element in the odd field

		// the original indices in the original 2D field
		int i2 = 2 * i + 1;
		int j2 = 2 * j + 1;

		int ido = i2*wo + j2; // the ID of the element in the original field

		// Each thread gets same seed, a different sequence number, no offset
		hiprand_init(seed, ido, 0, &states[id]);
	}
}

// 20170119
// setup the random states for all the elements in the 2D field
__global__ void 
kernel_setup_randstates_2d(hiprandState * states,	// output:the array of all the random states
						   int w, int h,			// input: the width, height of the 2D field
						   unsigned long long seed	// input: the seed
						   )
{
	int i = blockDim.y*blockIdx.y + threadIdx.y;
	int j = blockDim.x*blockIdx.x + threadIdx.x;

	if (i < h && j < w)
	{
		int id = i*w + j; // the ID of the element

		// Each thread gets same seed, a different sequence number, no offset
		hiprand_init(seed, id, 0, &states[id]);
	}
}

// 20170115
// uniform randomly drawn initialize a 2D array
__global__ void 
kernel_randinit_uniform_double(double * A,			// output:the array
							   hiprandState * states,// output:the random states
							   int w, int h,		// input: the width and height of the 2D array
							   double valmin,		// input: the minimum value
							   double valmax		// input: the maximum value
							   )
{
	int i = blockDim.y*blockIdx.y + threadIdx.y;
	int j = blockDim.x*blockIdx.x + threadIdx.x;

	if (i < h && j < w)
	{
		int id = i*w + j;

		// Copy state to local memory for efficiency
		hiprandState localState = states[id];

		double sample = hiprand_uniform_double(&localState);

		A[id] = (valmax - valmin)*sample + valmin;

		// Copy state back to global memory (update the state)
		states[id] = localState;
	}
}

// 20170115
// initilize the depth maps, alpha maps and beta maps using random numbers
__global__ void 
kernel_PatchMatch_randinit(hiprandState * states,			// output:the random states
						   double * depth,					// output:the depth map
						   double * alpha,					// output:the alpha map
						   double * beta,					// output:the beta map
						   int w, int h,					// input: the width and height of the maps
						   double min_d, double max_d,		// input: the minimum and maximum depth
						   double min_a, double max_a,		// input: the minimum and maximum alpha
						   double min_b, double max_b		// input: the minimum and maximum beta
						   )
{
	extern __shared__ unsigned char s_rect[]; // 20170319, the shared memory, adjacent pixels in base image share parts in support window

	int i = blockDim.y*blockIdx.y + threadIdx.y;
	int j = blockDim.x*blockIdx.x + threadIdx.x;

	if (i < h && j < w)
	{
		int id = i*w + j;

		// Copy state to local memory for efficiency
		hiprandState localState = states[id];

		double sample_depth = hiprand_uniform_double(&localState);
		double sample_alpha = hiprand_uniform_double(&localState);
		double sample_beta  = hiprand_uniform_double(&localState);

		depth[id] = (max_d - min_d)*sample_depth + min_d;
		alpha[id] = (max_a - min_a)*sample_alpha + min_a;
		beta[id]  = (max_b - min_b)*sample_beta  + min_b;

		// Copy state back to global memory (update the state)
		states[id] = localState;
	}
}

// 20170115
// initilize the depth maps, alpha maps and beta maps using random numbers
//__global__ void 
//kernel_PatchMatch_randinit(curandState * states,			// output:the random states
//						   double * depth,					// output:the depth map
//						   double * alpha,					// output:the alpha map
//						   double * beta,					// output:the beta map
//						   int w, int h,					// input: the width and height of the maps
//						   double min_d, double max_d,		// input: the minimum and maximum depth
//						   double min_a, double max_a,		// input: the minimum and maximum alpha
//						   double min_b, double max_b		// input: the minimum and maximum beta
//						   )
//{
//	int i = blockDim.y*blockIdx.y + threadIdx.y;
//	int j = blockDim.x*blockIdx.x + threadIdx.x;
//
//	if (i < h && j < w)
//	{
//		int id = i*w + j;
//
//		// Copy state to local memory for efficiency
//		curandState localState = states[id];
//
//		double sample_depth = curand_uniform_double(&localState);
//		double sample_alpha = curand_uniform_double(&localState);
//		double sample_beta  = curand_uniform_double(&localState);
//
//		depth[id] = (max_d - min_d)*sample_depth + min_d;
//		alpha[id] = (max_a - min_a)*sample_alpha + min_a;
//		beta[id]  = (max_b - min_b)*sample_beta  + min_b;
//
//		// Copy state back to global memory (update the state)
//		states[id] = localState;
//	}
//}

extern "C" void
CUDA_GenerateDSI_BT(int w, int h,				// input: the width and height of stereo images
				    const unsigned char * h_imgb,// input: h*w, the rectified grayscale base image
				    const unsigned char * h_imgm,// input: h*w, the rectified grayscale matching image
				    int dmin,					// input: scalar, the minimal disparity, can be negative, but make sure dmin<dmax
				    int dmax,					// input: scalar, the maximal disparity, can be negative, but make sure dmin<dmax
				    double * h_DSI,				// output:h*w*nd, nd = (dmax - dmin + 1), the output Disparity Space Image
				    int w_block,					// input: how many expected threads per row in a thread block
				    int h_block,					// input: how many expected threads per colomn in a thread block
				    int dir						// 0:l2r. 1:r2l. 2:u2b. 3:b2u
				    )
{
	const unsigned int nd = dmax - dmin + 1; // number of disparities

	const unsigned int nPixels = w*h; // number of pixels in total
	const unsigned int sizeMem = sizeof(unsigned char) * nPixels; // number of bytes of an image
	const unsigned int sizeMemDSI = sizeof(double) * nPixels * nd;// number of bytes of the DSI

	// allocate device memory
	unsigned char * d_imgb = NULL;
	unsigned char * d_imgm = NULL;
	double * d_DSI = NULL;
	hipMalloc((void **)&d_imgb, sizeMem);
	hipMalloc((void **)&d_imgm, sizeMem);
	hipMalloc((void **)&d_DSI, sizeMemDSI);

	// copy host memory to device memory
	hipMemcpy(d_imgb, h_imgb, sizeMem, hipMemcpyHostToDevice);
	hipMemcpy(d_imgm, h_imgm, sizeMem, hipMemcpyHostToDevice);
	hipMemcpy(d_DSI, h_DSI, sizeMemDSI, hipMemcpyHostToDevice);

	const unsigned int nBlocks_x = ((w % w_block) != 0) ? (w / w_block + 1) : (w / w_block); // number of blocks along x direction in the grid
	const unsigned int nBlocks_y = ((h % h_block) != 0) ? (h / h_block + 1) : (h / h_block); // number of blocks along y direction in the grid

	// determine the dimension of the grid and a block
	dim3 block(w_block, h_block, 1);
	dim3 grid(nBlocks_x, nBlocks_y, 1);

	/*cudaEvent_t start, stop;
	cudaEventCreate(&start);
	cudaEventCreate(&stop);

	cudaEventRecord(start, 0);*/

	// call cuda kernel
	if (0==dir) // l2r
	{
		kernel_GenerateDSI_l2r_BT<<<grid, block>>>(w, h, d_imgb, d_imgm, dmin, dmax, d_DSI);
	}
	else if (1==dir) // r2l
	{
		kernel_GenerateDSI_r2l_BT<<<grid, block>>>(w, h, d_imgb, d_imgm, dmin, dmax, d_DSI);
	}
	else if (2==dir) // u2b
	{
		kernel_GenerateDSI_u2b_BT<<<grid, block>>>(w, h, d_imgb, d_imgm, dmin, dmax, d_DSI);
	}
	else // b2u
	{
		kernel_GenerateDSI_b2u_BT<<<grid, block>>>(w, h, d_imgb, d_imgm, dmin, dmax, d_DSI);
	}

	/*cudaEventRecord(stop, 0);
	cudaEventSynchronize(stop);

	float elapsedTime = 0.0f;
	cudaEventElapsedTime(&elapsedTime, start, stop);*/

	hipMemcpy(h_DSI, d_DSI, sizeMemDSI, hipMemcpyDeviceToHost);

	hipFree(d_imgb);
	hipFree(d_imgm);
	hipFree(d_DSI);
}

extern "C" void
CUDA_GenerateDSI(int w, int h,					// input: the width and height of stereo images
				 int hw, int hh,				// input: the half width and half height of the support window
				 const unsigned char * h_imgb,	// input: h*w, the rectified grayscale base image
				 const unsigned char * h_imgm,	// input: h*w, the rectified grayscale matching image
				 int dmin,						// input: scalar, the minimal disparity, can be negative, but make sure dmin<dmax
				 int dmax,						// input: scalar, the maximal disparity, can be negative, but make sure dmin<dmax
				 double * h_DSI,				// output:h*w*nd, nd = (dmax - dmin + 1), the output Disparity Space Image
				 int w_block,					// input: how many expected threads per row in a thread block
				 int h_block,					// input: how many expected threads per colomn in a thread block
				 int dir,						// input: 0:l2r. 1:r2l. 2:u2b. 3:b2u
				 int costType                   // input: 0:BT,  1:ncc, 2:opencvncc
				 )
{
	//cudaDeviceProp deviceProp;
	//deviceProp.major = 0;
	//deviceProp.minor = 0;
	//int dev = 0;

	//int argc; char **argv;

	//// This will pick the best possible CUDA capable device
	//// Otherwise pick the device with highest Gflops/s
	//dev = gpuGetMaxGflopsDeviceId();
	//checkCudaErrors(cudaSetDevice(dev));
	//checkCudaErrors(cudaGetDeviceProperties(&deviceProp, dev));
	//printf("GPU Device %d: \"%s\" with compute capability %d.%d\n\n", dev, deviceProp.name, deviceProp.major, deviceProp.minor);

	//checkCudaErrors(cudaGetDeviceProperties(&deviceProp, dev));

	//// Statistics about the GPU device
	//printf("> GPU device has %d Multi-Processors, SM %d.%d compute capabilities\n\n",
	//	deviceProp.multiProcessorCount, deviceProp.major, deviceProp.minor);

	//int version = (deviceProp.major * 0x10 + deviceProp.minor);

	//if (version < 0x20)
	//{
	//	printf("requires a minimum CUDA compute 2.0 capability\n");
	//	exit(EXIT_SUCCESS);
	//}
	/////////////////////////////////////////////////////////////////////////////

	const unsigned int nd = dmax - dmin + 1; // number of disparities

	const unsigned int nPixels = w*h; // number of pixels in total
	const unsigned int sizeMem = sizeof(unsigned char) * nPixels; // number of bytes of an image
	const unsigned int sizeMemDSI = sizeof(double) * nPixels * nd;// number of bytes of the DSI

	// allocate device memory
	unsigned char * d_imgb = NULL;
	unsigned char * d_imgm = NULL;
	double * d_DSI = NULL;
	hipMalloc((void **)&d_imgb, sizeMem);
	hipMalloc((void **)&d_imgm, sizeMem);
	hipMalloc((void **)&d_DSI, sizeMemDSI);

	// copy host memory to device memory
	hipMemcpy(d_imgb, h_imgb, sizeMem, hipMemcpyHostToDevice);
	hipMemcpy(d_imgm, h_imgm, sizeMem, hipMemcpyHostToDevice);
	hipMemcpy(d_DSI, h_DSI, sizeMemDSI, hipMemcpyHostToDevice);

	const unsigned int nBlocks_x = ((w % w_block) != 0) ? (w / w_block + 1) : (w / w_block); // number of blocks along x direction in the grid
	const unsigned int nBlocks_y = ((h % h_block) != 0) ? (h / h_block + 1) : (h / h_block); // number of blocks along y direction in the grid

	// determine the dimension of the grid and a block
	dim3 block(w_block, h_block, 1);
	dim3 grid(nBlocks_x, nBlocks_y, 1);

//	// 20170101, try transpose ////////////////////////////////
//	unsigned char * d_imgb_t = NULL;
//	cudaMalloc((void **)&d_imgb_t, sizeMem);
//	forCUDA_ShowInfo("kernel_transpose starts");
//
//	cudaEvent_t start, stop;
//	cudaEventCreate(&start);
//	cudaEventCreate(&stop);
//
//	cudaEventRecord(start, 0);
//
//	for (int iii = 0; iii < 1000; ++iii)
//	{
////		kernel_transpose_uchar<<<grid, block>>>(w, h, d_imgb_t, d_imgb);
//		kernel_transpose_tile_uchar<<<grid, block, sizeof(unsigned char)*w_block*h_block>>>(w, h, d_imgb_t, d_imgb);
//	}
////	kernel_transpose_uchar<<<grid, block>>>(h, w, d_imgb, d_imgb_t);
//	kernel_transpose_tile_uchar<<<grid, block, sizeof(unsigned char)*w_block*h_block>>>(h, w, d_imgb, d_imgb_t);
//
//	cudaEventRecord(stop, 0);
//	cudaEventSynchronize(stop);
//
//	float elapsedTime = 0.0f;
//	cudaEventElapsedTime(&elapsedTime, start, stop);
//
//	forCUDA_ShowInfo("kernel_transpose ends");
//	cudaFree(d_imgb_t);
//	///////////////////////////////////////////////////////////

	/*cudaEvent_t start, stop;
	cudaEventCreate(&start);
	cudaEventCreate(&stop);

	cudaEventRecord(start, 0);*/

	forCUDA_ShowInfo("kernel_GenerateDSI starts");

	// call cuda kernel
//	kernel_GenerateDSI<<<grid, block>>>(w, h, hw, hh, d_imgb, d_imgm, dmin, dmax, d_DSI, dir, costType);
	kernel_GenerateDSI_new<<<grid, block>>>(w, h, hw, hh, d_imgb, d_imgm, dmin, dmax, d_DSI, dir, costType);

//	getLastCudaError("kernel_GenerateDSI execution failed\n");

	/*cudaError_t err = cudaGetLastError();

	const char * errorString = cudaGetErrorString(err);*/

	//CString strerr(errorString);
	//AfxMessageBox(strerr);
	
	/*if (cudaSuccess != err)
	{
		fprintf(stderr, "%s(%i) : getLastCudaError() CUDA error : %s : (%d) %s.\n",
			file, line, errorMessage, (int)err, cudaGetErrorString(err));
		DEVICE_RESET
			exit(EXIT_FAILURE);
	}*/
		
	/*cudaEventRecord(stop, 0);
	cudaEventSynchronize(stop);

	float elapsedTime = 0.0f;
	cudaEventElapsedTime(&elapsedTime, start, stop);*/

	hipMemcpy(h_DSI, d_DSI, sizeMemDSI, hipMemcpyDeviceToHost);

	forCUDA_ShowInfo("kernel_GenerateDSI ends");

	hipFree(d_imgb);
	hipFree(d_imgm);
	hipFree(d_DSI);
}

// 20170102, 3D thread block
extern "C" void
CUDA_GenerateDSI_new(int w, int h,					// input: the width and height of stereo images
					 int hw, int hh,				// input: the half width and half height of the support window
					 const unsigned char * h_imgb,	// input: h*w, the rectified grayscale base image
					 const unsigned char * h_imgm,	// input: h*w, the rectified grayscale matching image
					 int dmin,						// input: scalar, the minimal disparity, can be negative, but make sure dmin<dmax
					 int dmax,						// input: scalar, the maximal disparity, can be negative, but make sure dmin<dmax
					 double * h_DSI,				// output:h*w*nd, nd = (dmax - dmin + 1), the output Disparity Space Image
					 int w_block,					// input: how many expected threads per row in a thread block
					 int h_block,					// input: how many expected threads per colomn in a thread block
					 int k_block,					// input: how many expected threads per pillar in a thread block
					 int nTile,						// input: m, the transpose tile or thread block is m*m
					 int dir,						// input: 0:l2r. 1:r2l. 2:u2b. 3:b2u
					 int costType                   // input: 0:BT,  1:ncc, 2:opencvncc
					 )
{
	const unsigned int nd = dmax - dmin + 1; // number of disparities

	const unsigned int nPixels = w*h; // number of pixels in total
	const unsigned int sizeMem = sizeof(unsigned char) * nPixels; // number of bytes of an image
	const unsigned int sizeMemDSI = sizeof(double) * nPixels * nd;// number of bytes of the DSI

	// allocate device memory
	unsigned char * d_imgb = NULL;
	unsigned char * d_imgm = NULL;
	double * d_DSI = NULL;
	hipMalloc((void **)&d_imgb, sizeMem);
	hipMalloc((void **)&d_imgm, sizeMem);
	hipMalloc((void **)&d_DSI, sizeMemDSI);

	// copy host memory to device memory
	hipMemcpy(d_imgb, h_imgb, sizeMem, hipMemcpyHostToDevice);
	hipMemcpy(d_imgm, h_imgm, sizeMem, hipMemcpyHostToDevice);
	hipMemcpy(d_DSI, h_DSI, sizeMemDSI, hipMemcpyHostToDevice);

	const unsigned int nBlocks_x = ((w % w_block) != 0) ? (w / w_block + 1) : (w / w_block); // number of blocks along x direction in the grid
	const unsigned int nBlocks_y = ((h % h_block) != 0) ? (h / h_block + 1) : (h / h_block); // number of blocks along y direction in the grid
	const unsigned int nBlocks_z = ((nd % k_block) != 0) ? (nd / k_block + 1) : (nd / k_block); // number of blocks along z direction in the grid

	const unsigned int nBlocks_tile_x = ((w % nTile) != 0) ? (w / nTile + 1) : (w / nTile); // number of blocks along x direction in the grid
	const unsigned int nBlocks_tile_y = ((h % nTile) != 0) ? (h / nTile + 1) : (h / nTile); // number of blocks along y direction in the grid

	// determine the dimension of the 3D grid and 3D block
	dim3 block(k_block, w_block, h_block);
	dim3 grid(nBlocks_z, nBlocks_x, nBlocks_y);

	// determine the dimension of the 3D grid and 3D block if input stereo images are transposed
	dim3 block_t(k_block, h_block, w_block);
	dim3 grid_t(nBlocks_z, nBlocks_y, nBlocks_x);

	// determine the dimension of the 2D grid and 2D block for transpose kernel
	dim3 blockTile(nTile, nTile, 1);
	dim3 gridTile(nBlocks_tile_x, nBlocks_tile_y, 1);

	int sizeSharedMemTile = sizeof(unsigned char) * nTile * nTile;

	forCUDA_ShowInfo("CUDA_GenerateDSI_new starts");

	// call cuda kernel
	if (0 == dir) // l2r
	{
		if (0 == costType)
		{
			// BT
			kernel_GenerateDSI_l2r_BT<<<grid, block>>>(w, h, nd, d_imgb, d_imgm, dmin, d_DSI);
		}
		else if (1 == costType)
		{
			// ncc
			kernel_GenerateDSI_l2r_ncc<<<grid, block>>>(w, h, nd, hw, hh, d_imgb, d_imgm, dmin, d_DSI);
		}
		else
		{
			// opencv ncc
			kernel_GenerateDSI_l2r_opencvncc<<<grid, block>>>(w, h, nd, hw, hh, d_imgb, d_imgm, dmin, d_DSI);
		}
	}
	else if (1 == dir) // r2l
	{
		if (0 == costType)
		{
			// BT
			kernel_GenerateDSI_r2l_BT<<<grid, block>>>(w, h, nd, d_imgb, d_imgm, dmin, d_DSI);
		}
		else if (1 == costType)
		{
			// ncc
			kernel_GenerateDSI_r2l_ncc<<<grid, block>>>(w, h, nd, hw, hh, d_imgb, d_imgm, dmin, d_DSI);
		}
		else
		{
			// opencv ncc
			kernel_GenerateDSI_r2l_opencvncc<<<grid, block>>>(w, h, nd, hw, hh, d_imgb, d_imgm, dmin, d_DSI);
		}
	}
	else if (2 == dir) // u2b
	{
		unsigned char * d_imgb_t = NULL;
		unsigned char * d_imgm_t = NULL;
		hipMalloc((void **)&d_imgb_t, sizeMem);
		hipMalloc((void **)&d_imgm_t, sizeMem);

		// transpose both stereo images first
		kernel_transpose_tile_uchar<<<gridTile, blockTile, sizeSharedMemTile>>>(w, h, d_imgb_t, d_imgb);
		kernel_transpose_tile_uchar<<<gridTile, blockTile, sizeSharedMemTile>>>(w, h, d_imgm_t, d_imgm);

		if (0 == costType)
		{
			// BT
			kernel_GenerateDSI_u2b_BT_transposed<<<grid_t, block_t>>>(h, w, nd, d_imgb_t, d_imgm_t, dmin, d_DSI);
		}
		else if (1 == costType)
		{
			// ncc
			kernel_GenerateDSI_u2b_ncc_transposed<<<grid_t, block_t>>>(h, w, nd, hh, hw, d_imgb_t, d_imgm_t, dmin, d_DSI);
		}
		else
		{
			// opencv ncc
			kernel_GenerateDSI_u2b_opencvncc_transposed<<<grid_t, block_t>>>(h, w, nd, hh, hw, d_imgb_t, d_imgm_t, dmin, d_DSI);
		}

		hipFree(d_imgb_t);
		hipFree(d_imgm_t);
	}
	else // b2u
	{
		unsigned char * d_imgb_t = NULL;
		unsigned char * d_imgm_t = NULL;
		hipMalloc((void **)&d_imgb_t, sizeMem);
		hipMalloc((void **)&d_imgm_t, sizeMem);

		// transpose both stereo images first
		kernel_transpose_tile_uchar<<<gridTile, blockTile, sizeSharedMemTile>>>(w, h, d_imgb_t, d_imgb);
		kernel_transpose_tile_uchar<<<gridTile, blockTile, sizeSharedMemTile>>>(w, h, d_imgm_t, d_imgm);

		if (0 == costType)
		{
			// BT
			kernel_GenerateDSI_b2u_BT_transposed<<<grid_t, block_t>>>(h, w, nd, d_imgb_t, d_imgm_t, dmin, d_DSI);
		}
		else if (1 == costType)
		{
			// ncc
			kernel_GenerateDSI_b2u_ncc_transposed<<<grid_t, block_t>>>(h, w, nd, hh, hw, d_imgb_t, d_imgm_t, dmin, d_DSI);
		}
		else
		{
			// opencv ncc
			kernel_GenerateDSI_b2u_opencvncc_transposed<<<grid_t, block_t>>>(h, w, nd, hh, hw, d_imgb_t, d_imgm_t, dmin, d_DSI);
		}

		hipFree(d_imgb_t);
		hipFree(d_imgm_t);
	}

	hipMemcpy(h_DSI, d_DSI, sizeMemDSI, hipMemcpyDeviceToHost);

	forCUDA_ShowInfo("CUDA_GenerateDSI_new ends");

	hipFree(d_imgb);
	hipFree(d_imgm);
	hipFree(d_DSI);
}

extern "C" void
CUDA_CostAggregation_OneDir(int w, int h, int nd,		// input: the width and height of stereo images, and the number of disparities
							const double * h_DSI,		// input: h*w*nd, the Disparity Space Image
							double * h_C,				// output:h*w*nd, the aggregated cost volume along this direction
							double P1,					// input: constant penalty pixels in the neigborhood of (x,y), for which the disparity changes a little bit (that is 1 pixel)
							double P2,					// input: a larger constant penalty for all larger disparity changes
							int n_block,				// input: how many expected threads in a thread block
							int dir						// 0:l2r. 1:r2l. 2:u2b. 3:b2u
							)
{
//	cudaDeviceSetSharedMemConfig(cudaSharedMemBankSizeEightByte);

	const unsigned int sizeMemC = sizeof(double) *w * h * nd;// number of bytes of the cost volume

	// allocate device memory
	double * d_C = NULL;
	hipMalloc((void **)&d_C, sizeMemC);

	// copy host memory to device memory
	hipMemcpy(d_C, h_DSI, sizeMemC, hipMemcpyHostToDevice);

	const unsigned int nBlocks_x = ((w % n_block) != 0) ? (w / n_block + 1) : (w / n_block); // number of blocks along x direction in the grid
	const unsigned int nBlocks_y = ((h % n_block) != 0) ? (h / n_block + 1) : (h / n_block); // number of blocks along y direction in the grid

	int sizeMemPreC_block = sizeof(double) * nd * n_block;

	// determine the dimension of the grid and a block
	//dim3 block(w_block, h_block, 1);
	//dim3 grid(nBlocks_x, nBlocks_y, 1);

	//// call cuda kernel
	//kernel_GenerateDSI<<<grid, block>>>(w, h, hw, hh, d_imgb, d_imgm, dmin, dmax, d_DSI, dir, costType);

	forCUDA_ShowInfo("kernel_CostAggregation_OneParallelLine starts");

	dim3 block(n_block, 1, 1);

	if (0==dir) // up
	{
		dim3 grid(nBlocks_x, 1, 1);
		for (int i = 0; i < h; ++i)
		{
			kernel_CostAggregation_OneParallelLine<<<grid, block, sizeMemPreC_block>>>(w, h, nd, d_C, P1, P2, i, true, 0);
		}
	}
	else if (1==dir) // up right
	{
		if (w >= h) // using parallel row
		{
			dim3 grid(nBlocks_x, 1, 1);
			for (int i = 0; i < h; ++i)
			{
				kernel_CostAggregation_OneParallelLine<<<grid, block, sizeMemPreC_block>>>(w, h, nd, d_C, P1, P2, i, true, 1);
			}
		}
		else // using parallel colomn
		{
			dim3 grid(nBlocks_y, 1, 1);
			for (int j = (w - 1); j >= 0; --j)
			{
				kernel_CostAggregation_OneParallelLine<<<grid, block, sizeMemPreC_block>>>(w, h, nd, d_C, P1, P2, j, false, 1);
			}
		}
	}
	else if (2 == dir) // right
	{
		dim3 grid(nBlocks_y, 1, 1);
		for (int j = (w-1); j >= 0; --j)
		{
			kernel_CostAggregation_OneParallelLine<<<grid, block, sizeMemPreC_block>>>(w, h, nd, d_C, P1, P2, j, false, 2);
		}
	}
	else if (3 == dir) // bottom right
	{
		if (w >= h) // using parallel row
		{
			dim3 grid(nBlocks_x, 1, 1);
			for (int i = (h - 1); i >= 0; --i)
			{
				kernel_CostAggregation_OneParallelLine<<<grid, block, sizeMemPreC_block>>>(w, h, nd, d_C, P1, P2, i, true, 3);
			}
		}
		else // using parallel colomn
		{
			dim3 grid(nBlocks_y, 1, 1);
			for (int j = (w - 1); j >= 0; --j)
			{
				kernel_CostAggregation_OneParallelLine<<<grid, block, sizeMemPreC_block>>>(w, h, nd, d_C, P1, P2, j, false, 3);
			}
		}
	}
	else if (4 == dir) // bottom
	{
		dim3 grid(nBlocks_x, 1, 1);
		for (int i = (h-1); i >= 0; --i)
		{
			kernel_CostAggregation_OneParallelLine<<<grid, block, sizeMemPreC_block>>>(w, h, nd, d_C, P1, P2, i, true, 4);
		}
	}
	else if (5 == dir) // bottom left
	{
		if (w >= h) // using parallel row
		{
			dim3 grid(nBlocks_x, 1, 1);
			for (int i = (h - 1); i >= 0; --i)
			{
				kernel_CostAggregation_OneParallelLine<<<grid, block, sizeMemPreC_block>>>(w, h, nd, d_C, P1, P2, i, true, 5);
			}
		}
		else // using parallel colomn
		{
			dim3 grid(nBlocks_y, 1, 1);
			for (int j = 0; j < w; ++j)
			{
				kernel_CostAggregation_OneParallelLine<<<grid, block, sizeMemPreC_block>>>(w, h, nd, d_C, P1, P2, j, false, 5);
			}
		}
	}
	else if (6 == dir) // left
	{
		dim3 grid(nBlocks_y, 1, 1);
		for (int j = 0; j < w; ++j)
		{
			kernel_CostAggregation_OneParallelLine<<<grid, block, sizeMemPreC_block>>>(w, h, nd, d_C, P1, P2, j, false, 6);
		}
	}
	else if (7 == dir) // up left
	{
		if (w >= h)
		{
			dim3 grid(nBlocks_x, 1, 1);
			for (int i = 0; i < h; ++i)
			{
				kernel_CostAggregation_OneParallelLine<<<grid, block, sizeMemPreC_block>>>(w, h, nd, d_C, P1, P2, i, true, 7);
			}
		}
		else
		{
			dim3 grid(nBlocks_y, 1, 1);
			for (int j = 0; j < w; ++j)
			{
				kernel_CostAggregation_OneParallelLine<<<grid, block, sizeMemPreC_block>>>(w, h, nd, d_C, P1, P2, j, false, 7);
			}
		}
	}
	else
	{

	}

	hipMemcpy(h_C, d_C, sizeMemC, hipMemcpyDeviceToHost);

	forCUDA_ShowInfo("kernel_CostAggregation_OneParallelLine ends");

	hipFree(d_C);
}

extern "C" void
CUDA_CostAggregation_OneDir_new(int w, int h, int nd,		// input: the width and height of stereo images, and the number of disparities
								const double * h_DSI,		// input: h*w*nd, the Disparity Space Image
								double * h_C,				// output:h*w*nd, the aggregated cost volume along this direction
								double P1,					// input: constant penalty pixels in the neigborhood of (x,y), for which the disparity changes a little bit (that is 1 pixel)
								double P2,					// input: a larger constant penalty for all larger disparity changes
								int n_block,				// input: how many expected threads in a thread block
								int dir						// 0:l2r. 1:r2l. 2:u2b. 3:b2u
								)
{
//	cudaDeviceSetSharedMemConfig(cudaSharedMemBankSizeEightByte);

	const unsigned int sizeMemC = sizeof(double) *w * h * nd;// number of bytes of the cost volume

	// allocate device memory
	double * d_C = NULL;
	hipMalloc((void **)&d_C, sizeMemC);

	// copy host memory to device memory
	hipMemcpy(d_C, h_DSI, sizeMemC, hipMemcpyHostToDevice);

	const unsigned int nBlocks_x = ((w % n_block) != 0) ? (w / n_block + 1) : (w / n_block); // number of blocks along x direction in the grid
	const unsigned int nBlocks_y = ((h % n_block) != 0) ? (h / n_block + 1) : (h / n_block); // number of blocks along y direction in the grid

	// 20161228
	const unsigned int nBlocks_z = ((nd % n_block) != 0) ? (nd / n_block + 1) : (nd / n_block); // number of blocks along disparity direction in the grid

	forCUDA_ShowInfo("kernel_CostAggregation_OneParallelLine starts");

	// 1D parallelism
//	dim3 block(n_block, 1, 1);
//	dim3 grid_row(nBlocks_x, 1, 1);
//	dim3 grid_col(nBlocks_y, 1, 1);

	// 20161228, 2D parallelism
	dim3 block(n_block, n_block, 1);
	dim3 grid_row(nBlocks_z, nBlocks_x, 1);
	dim3 grid_col(nBlocks_z, nBlocks_y, 1);

	if (0==dir) // up
	{
		for (int i = 1; i < h; ++i)
		{
			kernel_CostAggregation_OneParallelRow_Up<<<grid_row, block>>>(w, h, nd, d_C, P1, P2, i);
		}
	}
	else if (1==dir) // up right
	{
		if (w >= h) // using parallel row
		{
			for (int i = 1; i < h; ++i)
			{
				kernel_CostAggregation_OneParallelRow_UpRight<<<grid_row, block>>>(w, h, nd, d_C, P1, P2, i);
			}
		}
		else // using parallel colomn
		{
			for (int j = (w - 2); j >= 0; --j)
			{
				kernel_CostAggregation_OneParallelCol_UpRight<<<grid_col, block>>>(w, h, nd, d_C, P1, P2, j);
			}
		}
	}
	else if (2 == dir) // right
	{
		for (int j = (w - 2); j >= 0; --j)
		{
			kernel_CostAggregation_OneParallelCol_Right<<<grid_col, block>>>(w, h, nd, d_C, P1, P2, j);
		}
	}
	else if (3 == dir) // bottom right
	{
		if (w >= h) // using parallel row
		{
			for (int i = (h - 2); i >= 0; --i)
			{
				kernel_CostAggregation_OneParallelRow_BottomRight<<<grid_row, block>>>(w, h, nd, d_C, P1, P2, i);
			}
		}
		else // using parallel colomn
		{
			for (int j = (w - 2); j >= 0; --j)
			{
				kernel_CostAggregation_OneParallelCol_BottomRight<<<grid_col, block>>>(w, h, nd, d_C, P1, P2, j);
			}
		}
	}
	else if (4 == dir) // bottom
	{
		for (int i = (h - 2); i >= 0; --i)
		{
			kernel_CostAggregation_OneParallelRow_Bottom<<<grid_row, block>>>(w, h, nd, d_C, P1, P2, i);
		}
	}
	else if (5 == dir) // bottom left
	{
		if (w >= h) // using parallel row
		{
			for (int i = (h - 2); i >= 0; --i)
			{
				kernel_CostAggregation_OneParallelRow_BottomLeft<<<grid_row, block>>>(w, h, nd, d_C, P1, P2, i);
			}
		}
		else // using parallel colomn
		{
			for (int j = 1; j < w; ++j)
			{
				kernel_CostAggregation_OneParallelCol_BottomLeft<<<grid_col, block>>>(w, h, nd, d_C, P1, P2, j);
			}
		}
	}
	else if (6 == dir) // left
	{
		for (int j = 1; j < w; ++j)
		{
			kernel_CostAggregation_OneParallelCol_Left<<<grid_col, block>>>(w, h, nd, d_C, P1, P2, j);
		}
	}
	else if (7 == dir) // up left
	{
		if (w >= h)
		{
			for (int i = 1; i < h; ++i)
			{
				kernel_CostAggregation_OneParallelRow_UpLeft<<<grid_row, block>>>(w, h, nd, d_C, P1, P2, i);
			}
		}
		else
		{
			for (int j = 1; j < w; ++j)
			{
				kernel_CostAggregation_OneParallelCol_UpLeft<<<grid_col, block>>>(w, h, nd, d_C, P1, P2, j);
			}
		}
	}
	else
	{

	}

	hipMemcpy(h_C, d_C, sizeMemC, hipMemcpyDeviceToHost);

	forCUDA_ShowInfo("kernel_CostAggregation_OneParallelLine ends");

	hipFree(d_C);
}

extern "C" void
CUDA_AddVec_double(double * h_A,		// input & output: A[i] += B[i]
				   const double * h_B,	// input: 
				   int n,				// input: number of elements
				   int nThreads			// input: number of threads in a thread block
				   )
{
	const unsigned int sizeMem = sizeof(double) * n;

	// allocate device memory
	double * d_A = NULL;
	double * d_B = NULL;
	hipMalloc((void **)&d_A, sizeMem);
	hipMalloc((void **)&d_B, sizeMem);

	// copy host memory to device memory
	hipMemcpy(d_A, h_A, sizeMem, hipMemcpyHostToDevice);
	hipMemcpy(d_B, h_B, sizeMem, hipMemcpyHostToDevice);

	const unsigned int nBlocks = ((n % nThreads) != 0) ? (n / nThreads + 1) : (n / nThreads); // number of blocks along x direction in the grid

	forCUDA_ShowInfo("kernel_AddVec_double starts");

	// determine the dimension of the grid and a block
	dim3 block(nThreads, 1, 1);
	dim3 grid(nBlocks, 1, 1);

	// run cuda kernel
	kernel_AddVec_double<<<grid, block>>>(d_A, d_B, n);

	hipMemcpy(h_A, d_A, sizeMem, hipMemcpyDeviceToHost);

	forCUDA_ShowInfo("kernel_AddVec_double ends");

	hipFree(d_A);
	hipFree(d_B);
}

extern "C" void
CUDA_transpose_uchar(unsigned char * h_B,		// output:B = A'
					 const unsigned char * h_A,	// input: A 
					 int w, int h,				// input: width and height of A
					 int nThreads				// input: m, the tile or thread block is m*m
					 )
{
	const unsigned int sizeMem = sizeof(unsigned char) * w * h;

	// allocate device memory
	unsigned char * d_A = NULL;
	unsigned char * d_B = NULL;
	hipMalloc((void **)&d_A, sizeMem);
	hipMalloc((void **)&d_B, sizeMem);

	// copy host memory to device memory
	hipMemcpy(d_A, h_A, sizeMem, hipMemcpyHostToDevice);

	const unsigned int nBlocks_x = ((w % nThreads) != 0) ? (w / nThreads + 1) : (w / nThreads); // number of blocks along x direction in the grid
	const unsigned int nBlocks_y = ((h % nThreads) != 0) ? (h / nThreads + 1) : (h / nThreads); // number of blocks along y direction in the grid

	// determine the dimension of the grid and a block
	dim3 block(nThreads, nThreads, 1);
	dim3 grid(nBlocks_x, nBlocks_y, 1);

	forCUDA_ShowInfo("kernel_transpose_uchar starts");
	
	// run cuda kernel
	kernel_transpose_uchar<<<grid, block>>>(w, h, d_B, d_A);

	hipMemcpy(h_B, d_B, sizeMem, hipMemcpyDeviceToHost);

	forCUDA_ShowInfo("kernel_transpose_uchar ends");

	hipFree(d_A);
	hipFree(d_B);
}

extern "C" void
CUDA_transpose_tile_uchar(unsigned char * h_B,			// output:B = A'
						  const unsigned char * h_A,	// input: A 
						  int w, int h,					// input: width and height of A
						  int nThreads					// input: m, the tile or thread block is m*m
						  )
{
	const unsigned int sizeMem = sizeof(unsigned char) * w * h;

	// allocate device memory
	unsigned char * d_A = NULL;
	unsigned char * d_B = NULL;
	hipMalloc((void **)&d_A, sizeMem);
	hipMalloc((void **)&d_B, sizeMem);

	// copy host memory to device memory
	hipMemcpy(d_A, h_A, sizeMem, hipMemcpyHostToDevice);

	const unsigned int nBlocks_x = ((w % nThreads) != 0) ? (w / nThreads + 1) : (w / nThreads); // number of blocks along x direction in the grid
	const unsigned int nBlocks_y = ((h % nThreads) != 0) ? (h / nThreads + 1) : (h / nThreads); // number of blocks along y direction in the grid

	// determine the dimension of the grid and a block
	dim3 block(nThreads, nThreads, 1);
	dim3 grid(nBlocks_x, nBlocks_y, 1);

	forCUDA_ShowInfo("kernel_transpose_tile_uchar starts");
	
	// run cuda kernel
	kernel_transpose_tile_uchar<<<grid, block>>>(w, h, d_B, d_A);

	hipMemcpy(h_B, d_B, sizeMem, hipMemcpyDeviceToHost);

	forCUDA_ShowInfo("kernel_transpose_tile_uchar ends");

	hipFree(d_A);
	hipFree(d_B);
}

// 20170104
//extern "C" void
//CUDA_PatchMatch(const unsigned char * h_imgb,	// input: the base gray image
//				const unsigned char * h_imgm,	// input: tha matching gray image
//				int w_b, int h_b,				// input: the width and height of base image
//				int w_m, int h_m,				// input: the width and height of matching image
//				double * h_depth,				// output:the estimated depth map of the base image
//				double * h_alpha,				// output:the estimated surface normal map of the base image
//				double * h_beta,				// output:the estimated surface normal map of the base image
//				int nThreads_w,					// input: the number of threads per row of the thread block
//				int nThreads_h,					// input: the number of threads per column of the thread block
//				unsigned long long randSeed,	// input: the random seed
//				double min_d, double max_d,		// input: the minimum and maximum depth
//				double min_a, double max_a,		// input: the minimum and maximum alpha
//				double min_b, double max_b		// input: the minimum and maximum beta
//				)
//{
//	int w_odd = w_b / 2; // the width of the odd maps
//	int h_odd = h_b / 2; // the height of the odd maps
//
//	int w_even = w_b - w_odd; // the width of the even maps
//	int h_even = h_b - h_odd; // the height of the even maps
//
//	const unsigned int sizeMemB = sizeof(unsigned char) * w_b * h_b;
//	const unsigned int sizeMemM = sizeof(unsigned char) * w_m * h_m;
//
//	const unsigned int sizeMemEven = sizeof(double) * w_even * h_even;
//	const unsigned int sizeMemOdd = sizeof(double) * w_odd * h_odd;
//
//	// allocate device memory
//	unsigned char * d_imgb = NULL;
//	unsigned char * d_imgm = NULL;
////	double * d_depth = NULL;
////	double * d_alpha = NULL;
////	double * d_beta  = NULL;
//	double * d_depth_even = NULL; double * d_depth_odd = NULL;
//	double * d_alpha_even = NULL; double * d_alpha_odd = NULL;
//	double * d_beta_even = NULL;  double * d_beta_odd = NULL;
//	cudaMalloc((void **)&d_imgb, sizeMemB);
//	cudaMalloc((void **)&d_imgm, sizeMemM);
//	cudaMalloc((void **)&d_depth_even, sizeMemEven);
//	cudaMalloc((void **)&d_depth_odd, sizeMemOdd);
//	cudaMalloc((void **)&d_alpha_even, sizeMemEven);
//	cudaMalloc((void **)&d_alpha_odd, sizeMemOdd);
//	cudaMalloc((void **)&d_beta_even, sizeMemEven);
//	cudaMalloc((void **)&d_beta_odd, sizeMemOdd);
//
//	// copy host memory to device memory
//	cudaMemcpy(d_imgb, h_imgb, sizeMemB, cudaMemcpyHostToDevice);
//	cudaMemcpy(d_imgm, h_imgm, sizeMemM, cudaMemcpyHostToDevice);
//
//	curandState * devStatesEven; // the curandStates of the even field
//	curandState * devStatesOdd;  // the curandStates of the odd field
//
//	// allocate the random states
//	cudaMalloc((void **)&devStatesEven, w_even * h_even * sizeof(curandState));
//	cudaMalloc((void **)&devStatesOdd, w_odd * h_odd * sizeof(curandState));
//
//	const unsigned int nBlocks_even_x = ((w_even % nThreads_w) != 0) ? (w_even / nThreads_w + 1) : (w_even / nThreads_w);
//	const unsigned int nBlocks_even_y = ((h_even % nThreads_h) != 0) ? (h_even / nThreads_h + 1) : (h_even / nThreads_h);
//
//	const unsigned int nBlocks_odd_x = ((w_odd % nThreads_w) != 0) ? (w_odd / nThreads_w + 1) : (w_odd / nThreads_w);
//	const unsigned int nBlocks_odd_y = ((h_odd % nThreads_h) != 0) ? (h_odd / nThreads_h + 1) : (h_odd / nThreads_h);
//
//	// determine the dimension of the grid and a block
//	dim3 block(nThreads_w, nThreads_h, 1);
//	dim3 grid_even(nBlocks_even_x, nBlocks_even_y, 1);
//	dim3 grid_odd(nBlocks_odd_x, nBlocks_odd_y, 1);
//
//	//double * h_depth_even = new double[w_even*h_even]; double * h_depth_odd = new double[w_odd * h_odd];
//	//double * h_alpha_even = new double[w_even*h_even]; double * h_alpha_odd = new double[w_odd * h_odd];
//	//double * h_beta_even = new double[w_even*h_even];  double * h_beta_odd = new double[w_odd * h_odd];
//
////	forCUDA_ShowInfo("random starts");
//
//	// setup the random states for the even and odd field
//	kernel_setup_randstates_even<<<grid_even, block>>>(devStatesEven, w_even, h_even, w_b, randSeed);
//	kernel_setup_randstates_odd<<<grid_odd, block>>>(devStatesOdd, w_odd, h_odd, w_b, randSeed);
//
//	// initialize all the parameter maps with uniform random parameters
//	kernel_PatchMatch_randinit<<<grid_even, block>>>(devStatesEven, d_depth_even, d_alpha_even, d_beta_even, w_even, h_even, min_d, max_d, min_a, max_a, min_b, max_b);
//	kernel_PatchMatch_randinit<<<grid_odd,  block>>>(devStatesOdd,  d_depth_odd,  d_alpha_odd,  d_beta_odd,  w_odd,  h_odd,  min_d, max_d, min_a, max_a, min_b, max_b);
//
//	/*kernel_PatchMatch_randinit<<<grid_even, block>>>(devStatesEven, d_depth_even, d_alpha_even, d_beta_even, w_even, h_even, 20, 100, 10, 360, 0, 60);
//	kernel_PatchMatch_randinit<<<grid_odd,  block>>>(devStatesOdd,  d_depth_odd,  d_alpha_odd,  d_beta_odd,  w_odd,  h_odd,  20, 100, 10, 360, 0, 60);
//
//	cudaMemcpy(h_depth_even, d_depth_even, sizeMemEven, cudaMemcpyDeviceToHost);
//	cudaMemcpy(h_alpha_even, d_alpha_even, sizeMemEven, cudaMemcpyDeviceToHost);
//	cudaMemcpy(h_beta_even, d_beta_even, sizeMemEven, cudaMemcpyDeviceToHost);
//	cudaMemcpy(h_depth_odd, d_depth_odd, sizeMemOdd, cudaMemcpyDeviceToHost);
//	cudaMemcpy(h_alpha_odd, d_alpha_odd, sizeMemOdd, cudaMemcpyDeviceToHost);
//	cudaMemcpy(h_beta_odd, d_beta_odd, sizeMemOdd, cudaMemcpyDeviceToHost);
//
//	forCUDA_ShowInfo("random ends");
//
//	forCUDA_SaveMatAsImage("E:\\results\\depth_even.png", h_depth_even, w_even, h_even, 20, 100);
//	forCUDA_SaveMatAsImage("E:\\results\\alpha_even.png", h_alpha_even, w_even, h_even, 10, 360);
//	forCUDA_SaveMatAsImage("E:\\results\\beta_even.png", h_beta_even, w_even, h_even, 0, 60);
//	forCUDA_SaveMatAsImage("E:\\results\\depth_odd.png", h_depth_odd, w_odd, h_odd, 20, 100);
//	forCUDA_SaveMatAsImage("E:\\results\\alpha_odd.png", h_alpha_odd, w_odd, h_odd, 10, 360);
//	forCUDA_SaveMatAsImage("E:\\results\\beta_odd.png", h_beta_odd, w_odd, h_odd, 0, 60);
//
//	delete[] h_depth_even;
//	delete[] h_alpha_even;
//	delete[] h_beta_even;
//	delete[] h_depth_odd;
//	delete[] h_alpha_odd;
//	delete[] h_beta_odd;*/
//
////	forCUDA_ShowInfo("kernel_transpose_tile_uchar starts");
//	
//	// run cuda kernel
////	kernel_transpose_tile_uchar<<<grid, block>>>(w, h, d_B, d_A);
//
////	cudaMemcpy(h_B, d_B, sizeMem, cudaMemcpyDeviceToHost);
//
////	forCUDA_ShowInfo("kernel_transpose_tile_uchar ends");
//
//	cudaFree(d_imgb);
//	cudaFree(d_imgm);
//	cudaFree(d_depth_even);
//	cudaFree(d_depth_odd);
//	cudaFree(d_alpha_even);
//	cudaFree(d_alpha_odd);
//	cudaFree(d_beta_even);
//	cudaFree(d_beta_odd);
//
//	cudaFree(devStatesEven);
//	cudaFree(devStatesOdd);
//}

// 20170119
extern "C" void
CUDA_PatchMatch(const unsigned char * h_imgb,	// input: the base gray image
				const unsigned char * h_imgm,	// input: tha matching gray image
				int w_b, int h_b,				// input: the width and height of base image
				int w_m, int h_m,				// input: the width and height of matching image
				double * h_depth,				// output:the estimated depth map of the base image
				double * h_alpha,				// output:the estimated surface normal map of the base image
				double * h_beta,				// output:the estimated surface normal map of the base image
				int nThreads_w,					// input: the number of threads per row of the thread block
				int nThreads_h,					// input: the number of threads per column of the thread block
				unsigned long long randSeed,	// input: the random seed
				double min_d, double max_d,		// input: the minimum and maximum depth
				double min_a, double max_a,		// input: the minimum and maximum alpha
				double min_b, double max_b		// input: the minimum and maximum beta
				)
{
	// determine the width and height of the parallel grid
	int w_grid = ((w_b % 2) != 0) ? ((w_b + 1) / 2) : (w_b / 2);
	int h_grid = h_b;

	int nPix = w_b*h_b;

	int sizeMemB = sizeof(unsigned char) * nPix;
	int sizeMemM = sizeof(unsigned char) * w_m * h_m;

	int sizeMemMaps = sizeof(double) * nPix;

	// allocate device memory
	unsigned char * d_imgb = NULL;
	unsigned char * d_imgm = NULL;
	double * d_depth = NULL;
	double * d_alpha = NULL;
	double * d_beta  = NULL;
	hipMalloc((void **)&d_imgb, sizeMemB);
	hipMalloc((void **)&d_imgm, sizeMemM);
	hipMalloc((void **)&d_depth, sizeMemMaps);
	hipMalloc((void **)&d_alpha, sizeMemMaps);
	hipMalloc((void **)&d_beta, sizeMemMaps);

	// copy host memory to device memory
	hipMemcpy(d_imgb, h_imgb, sizeMemB, hipMemcpyHostToDevice);
	hipMemcpy(d_imgm, h_imgm, sizeMemM, hipMemcpyHostToDevice);

	hiprandState * devStates; // the curandStates of the even field

	// allocate the random states
	hipMalloc((void **)&devStates, nPix * sizeof(hiprandState));

	int nBlocks_ox = ((w_b % nThreads_w) != 0) ? (w_b / nThreads_w + 1) : (w_b / nThreads_w);
	int nBlocks_x = ((w_grid % nThreads_w) != 0) ? (w_grid / nThreads_w + 1) : (w_grid / nThreads_w);
	int nBlocks_y = ((h_grid % nThreads_h) != 0) ? (h_grid / nThreads_h + 1) : (h_grid / nThreads_h);

	// determine the dimension of the grid and a block
	dim3 block(nThreads_w, nThreads_h, 1);
	dim3 grid(nBlocks_x, nBlocks_y, 1);
	dim3 grid_o(nBlocks_ox, nBlocks_y, 1);

	forCUDA_ShowInfo("PatchMatch starts");

	// setup the random states for all pixels
	kernel_setup_randstates_2d<<<grid_o, block>>>(devStates, w_b, h_b, randSeed);

	hipDeviceSynchronize();
	forCUDA_ShowInfo("1");

	// initialize all the parameter maps with uniform random parameters
	kernel_PatchMatch_randinit<<<grid_o, block>>>(devStates, d_depth, d_alpha, d_beta, w_b, h_b, min_d, max_d, min_a, max_a, min_b, max_b);

	hipDeviceSynchronize();
	forCUDA_ShowInfo("2");

	hipMemcpy(h_depth, d_depth, sizeMemMaps, hipMemcpyDeviceToHost);
	hipMemcpy(h_alpha, d_alpha, sizeMemMaps, hipMemcpyDeviceToHost);
	hipMemcpy(h_beta, d_beta, sizeMemMaps, hipMemcpyDeviceToHost);

	forCUDA_ShowInfo("PatchMatch ends");

	// save the results as images
	forCUDA_SaveMatAsImage("E:\\results\\depth.png", h_depth, w_b, h_b, min_d, max_d);
	forCUDA_SaveMatAsImage("E:\\results\\alpha.png", h_alpha, w_b, h_b, min_a, max_a);
	forCUDA_SaveMatAsImage("E:\\results\\beta.png", h_beta, w_b, h_b, min_b, max_b);
	
	hipFree(d_imgb);
	hipFree(d_imgm);
	hipFree(d_depth);
	hipFree(d_alpha);
	hipFree(d_beta);
	hipFree(devStates);
}

//}

